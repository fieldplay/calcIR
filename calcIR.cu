#include "hip/hip_runtime.h"
/*This is my first attempt to port my python ir program to cuda. 
 * It currently suffers from **very** slow excecution in python. 
 * I'm going to try to port it to cuda c */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <xdrfile/xdrfile.h>
#include <xdrfile/xdrfile_xtc.h>
#include "calcIR.h"
#include <complex.h>
#include <time.h>

#include "magma_v2.h"
#include <hipfft/hipfft.h>

int main(int argc, char *argv[])
{

    // ***              Variable Declaration            *** //
    // **************************************************** //

    // User input
    // TODO: make to get from user instead of hardcode
    const char          *gmxf         = argv[1];                                // trajectory file
    const char          *outf         = argv[2];
    const user_real_t   dt            = 0.010;                                  // dt between frames in xtc file (in ps)
    const int           ntcfpoints    = 100 ;                                   // the number of tcf points for each spectrum
    const int           nsamples      = 1   ;                                   // number of samples to average for the total spectrum
    const int           sampleEvery   = 5   ;                                   // sample a new configuration every sampleEvery ps. Note ntcfpoints*dt must be less than sampleEvery.

    const user_real_t   t1            = 0.260;                                  // relaxation time ( in ps )
    const user_real_t   avef          = 3415.2;                                 // the approximate average stretch frequency to get rid of high frequency oscillations in the time correlation function
    const int           omegaStart    = 2000;                                   // starting frequency for spectral density
    const int           omegaStop     = 5000;                                   // ending frequency for spectral density
    const int           omegaStep     = 5;                                      // resolution for spectral density

    const int           natom_mol     = 4;                                      // Atoms per water molecule  :: MODEL DEPENDENT
    const int           nchrom_mol    = 2;                                      // Chromophores per molecule :: TWO for stretch -- ONE for bend
    const int           nzeros        = 25600;                                  // zeros for padding fft
 


    // Some useful variables and constants
    int                 natoms, nmol, frame, nchrom;
    magma_int_t         nchrom2;
    const int           ntcfpointsR     = (nzeros + ntcfpoints - 1)*2;                  // number of points for the real fourier transform
    const int           nomega          = ( omegaStop - omegaStart ) / omegaStep + 1;   // number of frequencies for the spectral density
    int                 currentSample   = 0;                                            // current sample

    // Trajectory stuff for the CPU
    rvec                *x;                                                             // Position vector
    matrix              box;                                                            // Box vectors
    float               boxl, gmxtime, prec;                                            // Box lengths, time at current frame, precision of xtf file
    int                 step, xdrinfo;                                                  // The current step number

    // Some variables for the GPU
    rvec                *x_d;                                                           // positions
    user_real_t         *mux_d, *muy_d, *muz_d;                                         // transition dipole moments
    user_complex_t      *cmux0_d, *cmuy0_d, *cmuz0_d;                                   // complex version of the transition dipole moment at t=0 
    user_complex_t      *cmux_d, *cmuy_d, *cmuz_d;                                      // complex versions of the transition dipole moment
    user_complex_t      *tmpmu_d;                                                       // to sum all polarizations
    user_real_t         *MUX_d, *MUY_d, *MUZ_d;                                         // transition dipole moments in the eigen basis
    user_real_t         *eproj_d;                                                       // the electric field projected along the oh bonds
    user_real_t         *kappa_d;                                                       // the hamiltonian on the GPU
    const int           blockSize = 128;                                                // The number of threads to launch per block

    // magma variables for ssyevr
    user_real_t         aux_work[1];                                                    // To get optimal size of lwork
    magma_int_t         aux_iwork[1], info;                                             // To get optimal liwork, and return info
    magma_int_t         ldkappa, lwork, liwork;                                         // Leading dim of kappa, sizes of work arrays
    magma_int_t         *iwork;                                                         // Work array
    user_real_t         *work;                                                          // Work array
    user_real_t         *w   ;                                                          // Eigenvalues
    user_real_t         *wA  ;                                                          // Work array

    // magma variables for gemv
    magma_queue_t       queue;

    // variables for spectrum calculations
    user_real_t         *w_d;                                                           // Eigenvalues on the GPU
    user_real_t         *omega, *omega_d;                                               // Frequencies on CPU and GPU
    user_real_t         *Sw, *Sw_d;                                                     // Spectral density on CPU and GPU
    user_real_t         *tmpSw;                                                         // Temporary spectral density

    // variables for TCF
    user_complex_t      *F, *F_d;                                                       // F matrix on CPU and GPU
    user_complex_t      *prop, *prop_d;                                                 // Propigator matrix on CPU and GPU
    user_complex_t      *ctmpmat_d;                                                     // temporary complex matrix for matrix multiplications on gpu
    user_complex_t      *ckappa_d;                                                      // A complex version of kappa // TODO: CAN WE JUST CAST AS TYPE INSTEAD OF HAVING VARIABLES FOR THIS?
    user_complex_t      tcfx, tcfy, tcfz;                                               // Time correlation function, polarized
    user_complex_t      dcy, tcftmp;                                                    // Decay constant and a temporary variable for the tcf
    user_complex_t      *pdtcf, *pdtcf_d;                                               // padded time correlation functions
    user_complex_t      *tcf, *tcf_d;                                                   // Time correlation function
    user_complex_t      *tmptcf;                                                        // A temporary function for time correlation function
    user_real_t         *Ftcf, *Ftcf_d;                                                 // Fourier transformed time correlation function
    user_real_t         *tmpFtcf;                                                       // Temporary Fourier transformed time correlation function
    user_real_t         *time2;                                                         // Time array for tcf
    user_real_t         arg;                                                            // argument of exponential

    // For fft on gpu
    hipfftHandle         plan;

    // for timing
    time_t              start=time(NULL), end;

    // **************************************************** //
    // ***         End  Variable Declaration            *** //


    



    // ***          Begin main routine                  *** //
    // **************************************************** //

    // Open trajectory file and get info about the systeem

    XDRFILE *trj = xdrfile_open( gmxf, "r" ); 
    if ( trj == NULL )
    {
        printf("The file %s could not be opened. Is the name correct?\n", gmxf);
        exit(EXIT_FAILURE);
    }
    printf("Will read the trajectory from: %s.\n",gmxf);

    if ( argv[2] == NULL )
    {
        printf("No name was provided for output files. Please provide a name for the output files as the second argument\n");
        exit(EXIT_FAILURE);
    }


    read_xtc_natoms( (char *)gmxf, &natoms);
    nmol         = natoms / natom_mol;
    nchrom       = nmol * nchrom_mol;
    nchrom2      = (magma_int_t) nchrom*nchrom;
    ldkappa      = (magma_int_t) nchrom;

    printf("Found %d atoms and %d molecules.\n",natoms, nmol);
    printf("Found %d chromophores.\n",nchrom);


    // ***              MEMORY ALLOCATION               *** //
    // **************************************************** //

    // determine the number of blocks to launch on the gpu 
    // each thread takes care of one chromophore
    const int numBlocks = (nchrom+blockSize-1)/blockSize;
    
    // Initialize magma math library and initialize queue
    magma_init();
    magma_queue_create( 0, &queue ); 

    // allocate memory for arrays on the CPU
    x       = (rvec*)            malloc( natoms       * sizeof(x[0] ));
    omega   = (user_real_t *)    malloc( nomega       * sizeof(user_real_t));
    Sw      = (user_real_t *)    calloc( nomega       , sizeof(user_real_t));
    tmpSw   = (user_real_t *)    malloc( nomega       * sizeof(user_real_t));
    time2   = (user_real_t *)    malloc( ntcfpoints   * sizeof(user_real_t));
    Ftcf    = (user_real_t *)    calloc( ntcfpointsR  , sizeof(user_real_t));
    tmpFtcf = (user_real_t *)    malloc( ntcfpointsR  * sizeof(user_real_t));
    tmptcf  = (user_complex_t *) malloc( ntcfpoints   * sizeof(user_complex_t));
    tcf     = (user_complex_t *) calloc( ntcfpoints   , sizeof(user_complex_t));
    F       = (user_complex_t *) calloc( nchrom2      , sizeof(user_complex_t));
    prop    = (user_complex_t *) calloc( nchrom2      , sizeof(user_complex_t));


    
    // allocate memory for arrays on the GPU
    hipMalloc( &x_d     , natoms       *sizeof(x[0]));
    hipMalloc( &mux_d   , nchrom       *sizeof(user_real_t));
    hipMalloc( &muy_d   , nchrom       *sizeof(user_real_t));
    hipMalloc( &muz_d   , nchrom       *sizeof(user_real_t));
    hipMalloc( &MUX_d   , nchrom       *sizeof(user_real_t));
    hipMalloc( &MUY_d   , nchrom       *sizeof(user_real_t));
    hipMalloc( &MUZ_d   , nchrom       *sizeof(user_real_t));
    hipMalloc( &omega_d , nomega       *sizeof(user_real_t));
    hipMalloc( &Sw_d    , nomega       *sizeof(user_real_t));
    hipMalloc( &Ftcf_d  , ntcfpointsR  *sizeof(user_real_t));
    hipMalloc( &cmux_d  , nchrom       *sizeof(user_complex_t));
    hipMalloc( &cmuy_d  , nchrom       *sizeof(user_complex_t));
    hipMalloc( &cmuz_d  , nchrom       *sizeof(user_complex_t));
    hipMalloc( &cmux0_d , nchrom       *sizeof(user_complex_t));
    hipMalloc( &cmuy0_d , nchrom       *sizeof(user_complex_t));
    hipMalloc( &cmuz0_d , nchrom       *sizeof(user_complex_t));
    hipMalloc( &tmpmu_d , nchrom       *sizeof(user_complex_t));

#ifdef USE_DOUBLES
    magma_dmalloc( &eproj_d     , nchrom );
    magma_dmalloc( &kappa_d     , ldkappa*nchrom);
    magma_zmalloc( &ckappa_d    , nchrom2);
    magma_zmalloc( &F_d         , nchrom2);
    magma_zmalloc( &prop_d      , nchrom2);
    magma_zmalloc( &ctmpmat_d   , nchrom2);
    magma_dmalloc( &w_d         , nchrom );
    magma_zmalloc( &tcf_d       , ntcfpoints);
#else
    magma_smalloc( &eproj_d     , nchrom );
    magma_smalloc( &kappa_d     , ldkappa*nchrom);
    magma_cmalloc( &ckappa_d    , nchrom2);
    magma_cmalloc( &F_d         , nchrom2);
    magma_cmalloc( &prop_d      , nchrom2);
    magma_cmalloc( &ctmpmat_d   , nchrom2);
    magma_smalloc( &w_d         , nchrom );
    magma_cmalloc( &tcf_d       , ntcfpoints);
#endif


    // ***          END MEMORY ALLOCATION               *** //
    // **************************************************** //
    

    // **************************************************** //
    // ***          OUTER LOOP OVER SAMPLES             *** //

    while( currentSample < nsamples )
    {
        // search trajectory for current sample starting point
        xdrinfo = read_xtc( trj, natoms, &step, &gmxtime, box, x, &prec );
        if ( xdrinfo != 0 )
        {
            printf("WARNING:: read_xtc returned error %d.\nIs the trajectory long enough?\n", xdrinfo);
            exit(0);
        }
        if ( currentSample * sampleEvery == (int) gmxtime )
        {
            printf("Now processing sample %d starting at %.2f ps\n", currentSample, gmxtime );


        // **************************************************** //
        // ***         MAIN LOOP OVER TRAJECTORY            *** //
        for ( frame = 0; frame < ntcfpoints; frame++ )
        {



            // ---------------------------------------------------- //
            // ***          Get Info About The System           *** //

            // read the current frame from the trajectory file and copy to device memory
            // note it was read in the outer loop if we are at frame 0
            // also assume a square box, but this will need to be changed if it is not the case
            if ( frame != 0 ){
                read_xtc( trj, natoms, &step, &gmxtime, box, x, &prec );
            }
            hipMemcpy( x_d, x, natoms*sizeof(x[0]), hipMemcpyHostToDevice );
            boxl = box[0][0];

            // launch kernel to calculate the electric field projection along OH bonds and build the exciton hamiltonian
            get_eproj_GPU <<<numBlocks,blockSize>>> ( x_d, boxl, natoms, natom_mol, nchrom, nchrom_mol, nmol, eproj_d );

            get_kappa_GPU <<<numBlocks,blockSize>>> ( x_d, boxl, natoms, natom_mol, nchrom, nchrom_mol, nmol, eproj_d, 
                                                      kappa_d, mux_d, muy_d, muz_d );

            

            // ***          Done getting System Info            *** //
            // ---------------------------------------------------- //




            // ---------------------------------------------------- //
            // ***          Diagonalize the Hamiltonian         *** //

            // if the first time, query for optimal workspace dimensions
            if ( frame == 0 && currentSample == 0)
            {
#ifdef USE_DOUBLES
                magma_dsyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, NULL, ldkappa, 
                                  NULL, NULL, (magma_int_t) nchrom, aux_work, -1, aux_iwork, -1, &info );
#else
                magma_ssyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, NULL, ldkappa, 
                                  NULL, NULL, (magma_int_t) nchrom, aux_work, -1, aux_iwork, -1, &info );
#endif

                lwork   = (magma_int_t) aux_work[0];
                liwork  = aux_iwork[0];

                // make space for work arrays and eigenvalues and other stuff
#ifdef USE_DOUBLES
                magma_dmalloc_cpu   ( &w  , (magma_int_t) nchrom );
                magma_dmalloc_pinned( &wA , (magma_int_t) nchrom*ldkappa );
                magma_dmalloc_pinned( &work , lwork  );
                magma_imalloc_cpu   ( &iwork, liwork );
#else
                magma_smalloc_cpu   ( &w  , (magma_int_t) nchrom );
                magma_smalloc_pinned( &wA , (magma_int_t) nchrom*ldkappa );
                magma_smalloc_pinned( &work , lwork  );
                magma_imalloc_cpu   ( &iwork, liwork );
#endif
            }
#ifdef USE_DOUBLES
            magma_dsyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, kappa_d, ldkappa,
                              w, wA, ldkappa, work, lwork, iwork, liwork, &info );
#else
            magma_ssyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, kappa_d, ldkappa,
                              w, wA, ldkappa, work, lwork, iwork, liwork, &info );
#endif


            // ***          Done with the Diagonalization       *** //
            // ---------------------------------------------------- //



            // ---------------------------------------------------- //
            // ***              The Spectral Density            *** //

            if ( frame == 0 ){

                // project the transition dipole moments onto the eigenbasis
                // MU_d = kappa_d**T x mu_d 
#ifdef USE_DOUBLES
                magma_dgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, ldkappa, mux_d, 1, 0.0, MUX_d, 1, queue);
                magma_dgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, ldkappa, muy_d, 1, 0.0, MUY_d, 1, queue);
                magma_dgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, ldkappa, muz_d, 1, 0.0, MUZ_d, 1, queue);
#else
                magma_sgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, ldkappa, mux_d, 1, 0.0, MUX_d, 1, queue);
                magma_sgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, ldkappa, muy_d, 1, 0.0, MUY_d, 1, queue);
                magma_sgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, ldkappa, muz_d, 1, 0.0, MUZ_d, 1, queue);
#endif

                // Define the spectral range of interest and initialize spectral arrays
                for (int i = 0; i < nomega; i++)
                {
                    omega[i] = (user_real_t) (omegaStart + omegaStep*i); 
                    tmpSw[i] = 0.0;
                }

                // Copy relevant variables to device memory
                hipMemcpy( omega_d, omega, nomega*sizeof(user_real_t), hipMemcpyHostToDevice );
                hipMemcpy( w_d    , w    , nchrom*sizeof(user_real_t), hipMemcpyHostToDevice );
                hipMemcpy( Sw_d   , tmpSw, nomega*sizeof(user_real_t), hipMemcpyHostToDevice );

                // calculate the spectral density on the GPU and copy back to the CPU
                get_spectral_density <<<numBlocks,blockSize>>> ( w_d, MUX_d, MUY_d, MUZ_d, omega_d, Sw_d, nomega, nchrom, t1 );
                hipMemcpy( tmpSw, Sw_d, nomega*sizeof(user_real_t), hipMemcpyDeviceToHost );

                // Copy temporary to persistant to get average spectral density over samples
                for (int i = 0; i < nomega; i++ )
                {
                    Sw[i] += tmpSw[i];
                }
            }

            // ***           Done the Spectral Density          *** //
            // ---------------------------------------------------- //



            // ---------------------------------------------------- //

            // ***           Time Correlation Function          *** //

            // cast variables to complex to calculate time correlation function (which is complex)
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( kappa_d, ckappa_d, nchrom2);
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( mux_d  , cmux_d  , nchrom        );
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muy_d  , cmuy_d  , nchrom        );
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muz_d  , cmuz_d  , nchrom        );

            // First calculate the propigation matrix in the local basis
            if ( frame == 0 )
            {
                // initialize the F matrix at t=0 to the unit matrix
                for ( int i = 0; i < nchrom; i ++ )
                {
                    for ( int j = 0; j < nchrom; j ++ )
                    {
                        F[ i*nchrom + j] = MAGMA_ZERO;
                    }
                    F[ i*nchrom + i] = MAGMA_ONE;
                }
                // copy the F matrix to device memory -- after initialization, won't need back in host memory
                hipMemcpy( F_d, F, nchrom2*sizeof(user_complex_t), hipMemcpyHostToDevice );

                // set the transition dipole moment at t=0
                cast_to_complex_GPU <<<numBlocks,blockSize>>> ( mux_d  , cmux0_d  , nchrom );
                cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muy_d  , cmuy0_d  , nchrom );
                cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muz_d  , cmuz0_d  , nchrom );
            }
            else
            {
                // build the propigator
                for ( int i = 0; i < nchrom; i++ )
                {
                    // zero matrix
                    for ( int j = 0; j < nchrom; j ++ )
                    {
                        prop[ i*nchrom + j] = MAGMA_ZERO;
                    }
                    // P = exp(iwt/hbar)
                    arg   = ((w[i] - avef)* dt / HBAR);
                    prop[ i*nchrom + i ] = MAGMA_MAKE( cos(arg), sin(arg) );
                }

                // copy the propigator to the gpu and convert to the local basis
                // 
                hipMemcpy( prop_d, prop, nchrom2*sizeof(user_complex_t), hipMemcpyHostToDevice );

#ifdef USE_DOUBLES
                // ctmpmat_d = ckappa_d * prop_d
                magma_zgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             (magma_int_t) nchrom, MAGMA_ONE, ckappa_d, ldkappa, prop_d, ldkappa,
                              MAGMA_ZERO, ctmpmat_d, ldkappa, queue );

                // prop_d = ctmpmat_d * ckappa_d **T 
                magma_zgemm( MagmaNoTrans, MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             (magma_int_t) nchrom, MAGMA_ONE, ctmpmat_d, ldkappa, ckappa_d, ldkappa, 
                             MAGMA_ZERO, prop_d, ldkappa, queue );

                // ctmpmat_d = prop_d * F
                magma_zgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             (magma_int_t) nchrom, MAGMA_ONE, prop_d, ldkappa, F_d, ldkappa, 
                             MAGMA_ZERO, ctmpmat_d, ldkappa, queue );
#else
                // ctmpmat_d = ckappa_d * prop_d
                magma_cgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             (magma_int_t) nchrom, MAGMA_ONE, ckappa_d, ldkappa, prop_d, ldkappa,
                              MAGMA_ZERO, ctmpmat_d, ldkappa, queue );

                // prop_d = ctmpmat_d * ckappa_d **T 
                magma_cgemm( MagmaNoTrans, MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             (magma_int_t) nchrom, MAGMA_ONE, ctmpmat_d, ldkappa, ckappa_d, ldkappa, 
                             MAGMA_ZERO, prop_d, ldkappa, queue );

                // ctmpmat_d = prop_d * F
                magma_cgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             (magma_int_t) nchrom, MAGMA_ONE, prop_d, ldkappa, F_d, ldkappa, 
                             MAGMA_ZERO, ctmpmat_d, ldkappa, queue );
#endif

                // copy the F matrix back from the temporary variable to F_d
                copy_complex_GPU <<<numBlocks,blockSize>>> ( F_d  , ctmpmat_d  , nchrom2);
            }


            // calculate mFm for x y and z components
            // tcfx = cmux0_d**T * F_d *cmux_d
#ifdef USE_DOUBLES
            // x
            magma_zgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_ONE, F_d, ldkappa,
                         cmux0_d, 1, MAGMA_ZERO, tmpmu_d, 1, queue);
            tcfx = magma_zdotu( (magma_int_t) nchrom, cmux_d, 1, tmpmu_d, 1, queue );

            // y
            magma_zgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_ONE, F_d, ldkappa,
                         cmuy0_d, 1, MAGMA_ZERO, tmpmu_d, 1, queue);
            tcfy = magma_zdotu( (magma_int_t) nchrom, cmuy_d, 1, tmpmu_d, 1, queue );

            // z
            magma_zgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_ONE, F_d, ldkappa,
                         cmuz0_d, 1, MAGMA_ZERO, tmpmu_d, 1, queue);
            tcfz = magma_zdotu( (magma_int_t) nchrom, cmuz_d, 1, tmpmu_d, 1, queue );
#else
            // x
            magma_cgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_C_ONE, F_d, ldkappa,
                         cmux0_d, 1, MAGMA_C_ZERO, tmpmu_d, 1, queue);
            tcfx = magma_cdotu( (magma_int_t) nchrom, cmux_d, 1, tmpmu_d, 1, queue );

            // y
            magma_cgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_C_ONE, F_d, ldkappa,
                         cmuy0_d, 1, MAGMA_C_ZERO, tmpmu_d, 1, queue);
            tcfy = magma_cdotu( (magma_int_t) nchrom, cmuy_d, 1, tmpmu_d, 1, queue );

            // z
            magma_cgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_C_ONE, F_d, ldkappa,
                         cmuz0_d, 1, MAGMA_C_ZERO, tmpmu_d, 1, queue);
            tcfz = magma_cdotu( (magma_int_t) nchrom, cmuz_d, 1, tmpmu_d, 1, queue );
#endif

            // save the variables to print later and multiply by the decay
            time2[frame]          = dt * frame;
            tcftmp                = MAGMA_ADD( tcfx  , tcfy );
            tcftmp                = MAGMA_ADD( tcftmp, tcfz );
            dcy                   = MAGMA_MAKE(exp( -1.0 * frame * dt / ( 2.0 * t1 )), 0.0);
            tmptcf[frame]         = MAGMA_MUL( tcftmp, dcy );

 
            // ***        Done with Time Correlation            *** //
            // ---------------------------------------------------- //

        }

        // copy time correlation function to persistant memory to calculate average spectrum
        for ( int i = 0; i < ntcfpoints; i ++ )
        {
            tcf[i]  = MAGMA_ADD( tcf[i] , tmptcf[i]);
        }

        // done with current sample, move to next
        currentSample +=1;
        }
    } // end outer loop


    // close xdr file
    xdrfile_close(trj);


    // pad the time correlation function with zeros, copy to device memory and perform fft
    // fourier transform the time correlation function on the GPU
    pdtcf = (user_complex_t *) calloc( ntcfpoints+nzeros, sizeof(user_complex_t));
    for ( int i = 0; i < ntcfpoints; i++ )
    {
        pdtcf[i] = tcf[i];
    }
    for ( int i = 0; i < nzeros; i++ )
    {
        pdtcf[i+ntcfpoints] = MAGMA_ZERO;
    }

#ifdef USE_DOUBLES
    magma_zmalloc( &pdtcf_d    , ntcfpoints+nzeros);
    hipMemcpy   ( pdtcf_d, pdtcf, (ntcfpoints+nzeros)*sizeof(user_complex_t), hipMemcpyHostToDevice );
    hipfftPlan1d  ( &plan, ntcfpoints+nzeros, HIPFFT_Z2D, 1);
    hipfftExecZ2D ( plan, pdtcf_d, Ftcf_d );
#else
    magma_cmalloc( &pdtcf_d    , ntcfpoints+nzeros);
    hipMemcpy   ( pdtcf_d, pdtcf, (ntcfpoints+nzeros)*sizeof(user_complex_t), hipMemcpyHostToDevice );
    hipfftPlan1d  ( &plan, ntcfpoints+nzeros, HIPFFT_C2R, 1);
    hipfftExecC2R ( plan, pdtcf_d, Ftcf_d );
#endif
    hipMemcpy   ( Ftcf, Ftcf_d, ntcfpointsR*sizeof(user_real_t), hipMemcpyDeviceToHost );
    hipfftDestroy(plan);


    // normalize spectra by number of samples
    for ( int i = 0; i < ntcfpoints; i++ )
    {
        Ftcf[i] = Ftcf[i] / (user_real_t) nsamples; 
        tcf[i]  = MAGMA_DIV( tcf[i] , MAGMA_MAKE( nsamples, 0.0 ));
    }
    for ( int i = 0; i < nomega; i++)
    {
        Sw[i]   = Sw[i] / (user_real_t) nsamples;
    }

    // set base name for output files
    char * fname;
    fname = (char *) malloc( strlen(outf) + 9 );

    // write time correlation function
    FILE *rtcf = fopen(strcat(strcpy(fname,outf),"rtcf.dat"), "w");
    FILE *itcf = fopen(strcat(strcpy(fname,outf),"itcf.dat"), "w");
    for ( int i = 0; i < ntcfpoints; i++ )
    {
        fprintf( rtcf, "%g %g \n", time2[i], MAGMA_REAL( tcf[i] ) );
        fprintf( itcf, "%g %g \n", time2[i], MAGMA_IMAG( tcf[i] ) );
    }
    fclose( rtcf );
    fclose( itcf );

    // write the spectral density to file
    FILE *spec_density = fopen(strcat(strcpy(fname,outf),"spdn.dat"), "w");
    for ( int i = 0; i < nomega; i++)
    {
        fprintf(spec_density, "%e %e\n", omega[i], Sw[i]);
    }
    fclose(spec_density);

    // Write the absorption lineshape... Since the C2R transform is inverse by default, the frequencies have to be negated
    // note if you need to compare with YICUN's code, divide Ftcf by 2
    FILE *spec_lineshape = fopen(strcat(strcpy(fname,outf),"spec.dat"),"w");
    user_real_t factor   = 2*PI*HBAR/(dt*(ntcfpoints+nzeros));          // conversion factor to give energy and correct intensity from FFT
    for ( int i = (ntcfpoints+nzeros)/2; i < ntcfpoints+nzeros; i++ )   // "negative" FFT frequencies
    {
        if ( -1*(i-ntcfpoints-nzeros)*factor + avef <= (user_real_t) omegaStop  )
        {
            fprintf(spec_lineshape, "%e %e\n", -1*(i-ntcfpoints-nzeros)*factor + avef, Ftcf[i]/(factor*(ntcfpoints+nzeros)));
        }
    }
    for ( int i = 0; i < ntcfpoints+nzeros / 2 ; i++)                   // "positive" FFT frequencies
    {
        if ( -1*i*factor + avef >= (user_real_t) omegaStart)
        {
            fprintf(spec_lineshape, "%e %e\n", -1*i*factor + avef, Ftcf[i]/(factor*(ntcfpoints+nzeros)));
        }
    }
    fclose(spec_lineshape);

    // free memory on the CPU and GPU and finalize magma library
    magma_queue_destroy( queue );

    free(x);
    free(omega);
    free(Sw);
    free(tmpSw);
    free(time2);
    free(Ftcf);
    free(tmpFtcf);
    free(tcf);
    free(F);
    free(prop);
    free(pdtcf);

    hipFree(x_d);
    hipFree(mux_d); 
    hipFree(muy_d);
    hipFree(muz_d);
    hipFree(MUX_d); 
    hipFree(MUY_d);
    hipFree(MUZ_d);
    hipFree(omega_d);
    hipFree(Sw_d);
    hipFree(Ftcf_d);
    hipFree(cmux_d); 
    hipFree(cmuy_d);
    hipFree(cmuz_d);
    hipFree(cmux0_d); 
    hipFree(cmuy0_d);
    hipFree(cmuz0_d);
    hipFree(tmpmu_d);
 
    magma_free(eproj_d);
    magma_free(kappa_d);
    magma_free(ckappa_d);
    magma_free(F_d);
    magma_free(prop_d);
    magma_free(ctmpmat_d);
    magma_free(w_d);
    magma_free(tcf_d);
    magma_free(pdtcf_d);

    magma_free_cpu(w);
    magma_free_cpu(iwork);
    magma_free_pinned( work );
    magma_free_pinned( wA );

    // final call to finalize magma math library
    magma_finalize();

    end = time(NULL);
    printf("Done with the calculation in %f seconds.\n", difftime(end,start));

    return 0;
}

/**********************************************************
   
   BUILD ELECTRIC FIELD PROJECTION ALONG OH BONDS
                    GPU FUNCTION

 **********************************************************/
__global__
void get_eproj_GPU( rvec *x, float boxl, int natoms, int natom_mol, int nchrom, int nchrom_mol, int nmol, user_real_t  *eproj )
{
    
    int n, m, i, j, istart, istride;
    int chrom;
    user_real_t mox[DIM];                     // oxygen position on molecule m
    user_real_t mx[DIM];                      // atom position on molecule m
    user_real_t nhx[DIM];                     // hydrogen position on molecule n of the current chromophore
    user_real_t nox[DIM];                     // oxygen position on molecule n
    user_real_t nohx[DIM];                    // the unit vector pointing along the OH bond for the current chromophore
    user_real_t dr[DIM];                      // the min image vector between two atoms
    user_real_t r;                            // the distance between two atoms 
    const float cutoff = 0.7831;         // the oh cutoff distance
    const float bohr_nm = 18.8973;       // convert from bohr to nanometer
    user_real_t efield[DIM];                  // the electric field vector

    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // Loop over the chromophores belonging to the current thread
    for ( chrom = istart; chrom < nchrom; chrom += istride )
    {
        // calculate the molecule hosting the current chromophore 
        n = chrom / nchrom_mol;

        // initialize the electric field vector to zero at this chromophore
        efield[0]   =   0.;
        efield[1]   =   0.;
        efield[2]   =   0.;


        // ***          GET INFO ABOUT MOLECULE N HOSTING CHROMOPHORE       *** //
        //                      N IS OUR REFERENCE MOLECULE                     //
        // get the position of the hydrogen associated with the current stretch 
        // NOTE: I'm making some assumptions about the ordering of the positions, 
        // this can be changed if necessary for a more robust program
        // Throughout, I assume that the atoms are grouped into molecules and that
        // every 4th molecule starting at 0 (1, 2, 3) is OW (HW1, HW2, MW)
        if ( chrom % 2 == 0 ){      //HW1
            nhx[0]  = x[ n*natom_mol + 1 ][0];
            nhx[1]  = x[ n*natom_mol + 1 ][1];
            nhx[2]  = x[ n*natom_mol + 1 ][2];
        }
        else if ( chrom % 2 == 1 ){ //HW2
            nhx[0]  = x[ n*natom_mol + 2 ][0];
            nhx[1]  = x[ n*natom_mol + 2 ][1];
            nhx[2]  = x[ n*natom_mol + 2 ][2];
        }

        // The oxygen position
        nox[0]  = x[ n*natom_mol ][0];
        nox[1]  = x[ n*natom_mol ][1];
        nox[2]  = x[ n*natom_mol ][2];

        // The oh unit vector
        nohx[0] = minImage( nhx[0] - nox[0], boxl );
        nohx[1] = minImage( nhx[1] - nox[1], boxl );
        nohx[2] = minImage( nhx[2] - nox[2], boxl );
        r       = mag3(nohx);
        nohx[0] /= r;
        nohx[1] /= r;
        nohx[2] /= r;
        // for testing with YICUN -- can change to ROH later...
        //nohx[0] /= 0.09572;
        //nohx[1] /= 0.09572;
        //nohx[2] /= 0.09572;
 
        // ***          DONE WITH MOLECULE N                                *** //



        // ***          LOOP OVER ALL OTHER MOLECULES                       *** //
        for ( m = 0; m < nmol; m++ ){

            // skip the reference molecule
            if ( m == n ) continue;

            // get oxygen position on current molecule
            mox[0] = x[ m*natom_mol ][0];
            mox[1] = x[ m*natom_mol ][1];
            mox[2] = x[ m*natom_mol ][2];

            // find displacement between oxygen on m and hydrogen on n
            dr[0]  = minImage( mox[0] - nhx[0], boxl );
            dr[1]  = minImage( mox[1] - nhx[1], boxl );
            dr[2]  = minImage( mox[2] - nhx[2], boxl );
            r      = mag3(dr);

            // skip if the distance is greater than the cutoff
            if ( r > cutoff ) continue;

            // loop over all atoms in the current molecule and calculate the electric field 
            // (excluding the oxygen atoms since they have no charge)
            for ( i=1; i < natom_mol; i++ ){

                // position of current atom
                mx[0] = x[ m*natom_mol + i ][0];
                mx[1] = x[ m*natom_mol + i ][1];
                mx[2] = x[ m*natom_mol + i ][2];

                // the minimum image displacement between the reference hydrogen and the current atom
                // NOTE: this converted to bohr so the efield will be in au
                dr[0]  = minImage( nhx[0] - mx[0], boxl )*bohr_nm;
                dr[1]  = minImage( nhx[1] - mx[1], boxl )*bohr_nm;
                dr[2]  = minImage( nhx[2] - mx[2], boxl )*bohr_nm;
                r      = mag3(dr);

                // Add the contribution of the current atom to the electric field
                if ( i < 3  ){              // HW1 and HW2
                    for ( j=0; j < DIM; j++){
                        efield[j] += 0.52 * dr[j] / (r*r*r);
                    }
                }
                else if ( i == 3 ){         // MW (note the negative sign)
                    for ( j=0; j < DIM; j++){
                        efield[j] -= 1.04 * dr[j] / (r*r*r);
                    }
                }
            } // end loop over atoms in molecule m

        } // end loop over molecules m

        // project the efield along the OH bond to get the relevant value for the map
        eproj[chrom] = dot3( efield, nohx );

        // test looks good, everything appears to be ok -- a little different than YICUN, but i think it is numerical error
        /*
        if( chrom == 0 ){
            printf("chrom %d En %g\n", chrom, eproj[chrom]);
            printf("%g %g %g\n", efield[0], efield[1], efield[2]);
            printf("%g %g %g\n", nohx[0], nohx[1], nohx[2]);
        }
        */


        //printf("chrom: %d, eproj %f \n", chrom, eproj[chrom]);

    } // end loop over reference chromophores
}

/**********************************************************
   
   BUILD HAMILTONIAN AND RETURN TRANSITION DIPOLE VECTOR
    FOR EACH CHROMOPHORE ON THE GPU

 **********************************************************/
__global__
void get_kappa_GPU( rvec *x, float boxl, int natoms, int natom_mol, int nchrom, int nchrom_mol, int nmol, 
                    user_real_t *eproj, user_real_t *kappa, user_real_t *mux, user_real_t *muy, user_real_t *muz)
{
    
    int n, m, istart, istride;
    int chromn, chromm;
    user_real_t mox[DIM];                         // oxygen position on molecule m
    user_real_t mhx[DIM];                         // atom position on molecule m
    user_real_t nhx[DIM];                         // hydrogen position on molecule n of the current chromophore
    user_real_t nox[DIM];                         // oxygen position on molecule n
    user_real_t noh[DIM];
    user_real_t moh[DIM];
    user_real_t nmu[DIM];
    user_real_t mmu[DIM];
    user_real_t mmuprime;
    user_real_t nmuprime;
    user_real_t dr[DIM];                          // the min image vector between two atoms
    user_real_t r;                                // the distance between two atoms 
    const user_real_t bohr_nm    = 18.8973;       // convert from bohr to nanometer
    const user_real_t cm_hartree = 2.1947463E5;   // convert from cm-1 to hartree
    user_real_t En, Em;                           // the electric field projection
    user_real_t xn, xm, pn, pm;                   // the x and p from the map
    user_real_t wn, wm;                           // the energies

    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // Loop over the chromophores belonging to the current thread and fill in kappa for that row
    for ( chromn = istart; chromn < nchrom; chromn += istride )
    {
        // calculate the molecule hosting the current chromophore 
        // and get the corresponding electric field at the relevant hydrogen
        n   = chromn / nchrom_mol;
        En  = eproj[chromn];

        // build the map
        wn  = 3760.2 - 3541.7*En - 152677.0*En*En;
        xn  = 0.19285 - 1.7261E-5 * wn;
        pn  = 1.6466  + 5.7692E-4 * wn;
        nmuprime = 0.1646 + 11.39*En + 63.41*En*En;

        // and calculate the location of the transition dipole moment
        // SEE calc_efield_GPU for assumptions about ordering of atoms
        nox[0]  = x[ n*natom_mol ][0];
        nox[1]  = x[ n*natom_mol ][1];
        nox[2]  = x[ n*natom_mol ][2];
        if ( chromn % 2 == 0 )       //HW1
        {
            nhx[0]  = x[ n*natom_mol + 1 ][0];
            nhx[1]  = x[ n*natom_mol + 1 ][1];
            nhx[2]  = x[ n*natom_mol + 1 ][2];
        }
        else if ( chromn % 2 == 1 )  //HW2
        {
            nhx[0]  = x[ n*natom_mol + 2 ][0];
            nhx[1]  = x[ n*natom_mol + 2 ][1];
            nhx[2]  = x[ n*natom_mol + 2 ][2];
        }

        // The OH unit vector
        noh[0] = minImage( nhx[0] - nox[0], boxl );
        noh[1] = minImage( nhx[1] - nox[1], boxl );
        noh[2] = minImage( nhx[2] - nox[2], boxl );
        r      = mag3(noh);
        noh[0] /= r;
        noh[1] /= r;
        noh[2] /= r;

        // The location of the TDM
        nmu[0] = minImage( nox[0] + 0.067 * noh[0], boxl );
        nmu[1] = minImage( nox[1] + 0.067 * noh[1], boxl );
        nmu[2] = minImage( nox[2] + 0.067 * noh[2], boxl );
        
        // and the TDM vector to return
        mux[chromn] = noh[0] * nmuprime * xn;
        muy[chromn] = noh[1] * nmuprime * xn;
        muz[chromn] = noh[2] * nmuprime * xn;



        // Loop over all other chromophores
        for ( chromm = 0; chromm < nchrom; chromm ++ )
        {
            // calculate the molecule hosting the current chromophore 
            // and get the corresponding electric field at the relevant hydrogen
            m   = chromm / nchrom_mol;
            Em  = eproj[chromm];

            // also get the relevent x and p from the map
            wm  = 3760.2 - 3541.7*Em - 152677.0*Em*Em;
            xm  = 0.19285 - 1.7261E-5 * wm;
            pm  = 1.6466  + 5.7692E-4 * wm;
            mmuprime = 0.1646 + 11.39*Em + 63.41*Em*Em;

            // the diagonal energy
            if ( chromn == chromm )
            {
                //kappa[chromn*nchrom + chromm]   =   3500.0;
                //continue;
                // Note that this is a flattened 2d array
                kappa[chromn*nchrom + chromm]   =   wm; 
            }

            // intramolecular coupling
            else if ( m == n )
            {
                //kappa[chromn*nchrom + chromm]   =   0.0;
                //continue;
                kappa[chromn*nchrom + chromm]   =  (-1361.0 + 27165*(En + Em))*xn*xm - 1.887*pn*pm;
            }

            // intermolecular coupling
            else
            {
                
                //kappa[chromn*nchrom + chromm]   =   0.0;
                //continue;
                
                // calculate the distance between dipoles
                // they are located 0.67 A from the oxygen along the OH bond
                // tdm position on chromophore n
                mox[0]  = x[ m*natom_mol ][0];
                mox[1]  = x[ m*natom_mol ][1];
                mox[2]  = x[ m*natom_mol ][2];
                if ( chromm % 2 == 0 )       //HW1
                {
                    mhx[0]  = x[ m*natom_mol + 1 ][0];
                    mhx[1]  = x[ m*natom_mol + 1 ][1];
                    mhx[2]  = x[ m*natom_mol + 1 ][2];
                }
                else if ( chromm % 2 == 1 )  //HW2
                {
                    mhx[0]  = x[ m*natom_mol + 2 ][0];
                    mhx[1]  = x[ m*natom_mol + 2 ][1];
                    mhx[2]  = x[ m*natom_mol + 2 ][2];
                }

                // The OH unit vector
                moh[0] = minImage( mhx[0] - mox[0], boxl );
                moh[1] = minImage( mhx[1] - mox[1], boxl );
                moh[2] = minImage( mhx[2] - mox[2], boxl );
                r      = mag3(moh);
                moh[0] /= r;
                moh[1] /= r;
                moh[2] /= r;

                // The location of the TDM and the dipole derivative
                mmu[0] = minImage( mox[0] + 0.067 * moh[0], boxl );
                mmu[1] = minImage( mox[1] + 0.067 * moh[1], boxl );
                mmu[2] = minImage( mox[2] + 0.067 * moh[2], boxl );

                // the distance between TDM on N and on M and convert to unit vector
                dr[0] = minImage( nmu[0] - mmu[0], boxl );
                dr[1] = minImage( nmu[1] - mmu[1], boxl );
                dr[2] = minImage( nmu[2] - mmu[2], boxl );
                r     = mag3( dr );
                dr[0] /= r;
                dr[1] /= r;
                dr[2] /= r;
                r     *= bohr_nm; // convert to bohr

                // The coupling in the transition dipole approximation in wavenumber
                // Note the conversion to wavenumber
                kappa[chromn*nchrom + chromm]   = ( dot3( noh, moh ) - 3.0 * dot3( noh, dr ) * 
                                                    dot3( moh, dr ) ) / ( r*r*r ) * 
                                                    xn*xm*nmuprime*mmuprime*cm_hartree;
            }// end intramolecular coupling
        }// end loop over chromm
    }// end loop over reference
}


/**********************************************************
   
        Calculate the Spectral Density

 **********************************************************/
__global__
void get_spectral_density( user_real_t *w, user_real_t *MUX, user_real_t *MUY, user_real_t *MUZ, user_real_t *omega, user_real_t *Sw, 
                           int nomega, int nchrom, user_real_t t1 ){

    int istart, istride, i, chromn;
    user_real_t wi, dw, MU2, gamma;
    
    // split up each desired frequency to separate thread on GPU
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // the linewidth parameter
    gamma = HBAR/(t1 * 2.0);

    // Loop over the chromophores belonging to the current thread and fill in kappa for that row
    for ( i = istart; i < nomega; i += istride )
    {
        // get current frequency
        wi = omega[i];
        
        // Loop over all chromophores calculatint the spectral intensity at the current frequency
        for ( chromn = 0; chromn < nchrom; chromn ++ ){
            // calculate the TDM squared and get the mode energy
            MU2     = MUX[chromn]*MUX[chromn] + MUY[chromn]*MUY[chromn] + MUZ[chromn]*MUZ[chromn];
            dw      = wi - w[chromn];

            // add a lorentzian lineshape to the spectral density
            Sw[i] += MU2 * gamma / ( dw*dw + gamma*gamma )/PI;
        }
    }
}

/**********************************************************
   
        HELPER FUNCTIONS FOR GPU CALCULATIONS
            CALLABLE FROM CPU AND GPU

 **********************************************************/



// The minimage image of a scalar
user_real_t minImage( user_real_t dx, user_real_t boxl )
{
    return dx - boxl*round(dx/boxl);
}



// The magnitude of a 3 dimensional vector
user_real_t mag3( user_real_t dx[3] )
{
    return sqrt( dot3( dx, dx ) );
}



// The dot product of a 3 dimensional vector
user_real_t dot3( user_real_t x[3], user_real_t y[3] )
{
    return  x[0]*y[0] + x[1]*y[1] + x[2]*y[2];
}



// cast the matrix from float to complex -- this may not be the best way to do this, but it is quick to implement
__global__
void cast_to_complex_GPU ( user_real_t *d_d, user_complex_t *z_d, magma_int_t n )
{
    int istart, istride, i;
    
    // split up each desired frequency to separate thread on GPU
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // convert from float to complex
    for ( i = istart; i < n; i += istride )
    {
        z_d[i] = MAGMA_MAKE( d_d[i], 0.0 ); 
    }
}



// copy a complex matrix to another one
__global__
void copy_complex_GPU( user_complex_t *out_d, user_complex_t *in_d, magma_int_t n )
{
    int istart, istride, i;
    
    // split up each desired frequency to separate thread on GPU
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // convert from float to complex
    for ( i = istart; i < n; i += istride )
    {
        out_d[i] = in_d[i];
    }
}
