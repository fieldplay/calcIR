#include "hip/hip_runtime.h"
/*This is my first attempt to port my python ir program to cuda. 
 * It currently suffers from **very** slow excecution in python. 
 * I'm going to try to port it to cuda c */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <xdrfile/xdrfile.h>
#include <xdrfile/xdrfile_xtc.h>
#include "calcIR.h"
#include <complex.h>

#include "magma_v2.h"
#include <hipfft/hipfft.h>

#define HBAR 5.308837367       // in cm-1 * ps
#define PI   3.14159265359


int main()
{

    // ***              Variable Declaration            *** //
    // **************************************************** //

    // User input
    // TODO: make to get from user instead of hardcode
    const char   *gmxf         = (const char *)"./n216/traj_comp.xtc"; // trajectory file
    const double dt            = 0.010;  // dt between frames in xtc file (in ps)
    const int    ntcfpoints    = 500 ;   // the number of tcf points for each spectrum
    const int    nsamples      = 1   ;   // number of samples to average for the total spectrum
    const int    sampleEvery   = 5   ;   // sample a new configuration every sampleEvery ps. Note ntcfpoints*dt must be less than sampleEvery.

    const double t1            = 0.260;  // relaxation time ( in ps )
    const double avef          = 3415.2; // the approximate average stretch frequency to get rid of high frequency oscillations in the time correlation function
    const int   omegaStart     = 2000;   // starting frequency for spectral density
    const int   omegaStop      = 5000;   // ending frequency for spectral density
    const int   omegaStep      = 5;      // resolution for spectral density

    const int   natom_mol      = 4;      // Atoms per water molecule  :: MODEL DEPENDENT
    const int   nchrom_mol     = 2;      // Chromophores per molecule :: TWO for stretch -- ONE for bend
    
    const int   nzeros         = 25600;  // zeros for padding fft -- what was used by Yicun
 


    // Some useful variables and constants
    int               natoms, nmol, frame, nchrom;
    const int         ntcfpointsR = (nzeros + ntcfpoints - 1)*2;                         // number of points for the real fourier transform
    const int         nomega      = ( omegaStop - omegaStart ) / omegaStep + 1; // number of frequencies for the spectral density
    int               currentSample = 0;                                        // current sample

    // Trajectory stuff for the CPU
    rvec        *x;                     // Position vector
    matrix      box;                    // Box vectors
    float       boxl, gmxtime, prec;    // Box lengths, time at current frame, precision of xtf file
    int         step, xdrinfo;          // The current step number

    // Some variables for the GPU
    rvec                *x_d;                       // positions
    double              *mux_d, *muy_d, *muz_d;     // transition dipole moments
    magmaDoubleComplex  *cmux0_d, *cmuy0_d, *cmuz0_d;// complex version of the transition dipole moment at t=0 
    magmaDoubleComplex  *cmux_d, *cmuy_d, *cmuz_d;  // complex versions of the transition dipole moment
    magmaDoubleComplex  *tmpmu_d;                   // to sum all polarizations
    double              *MUX_d, *MUY_d, *MUZ_d;     // transition dipole moments in the eigen basis
    double              *eproj_d;                   // the electric field projected along the oh bonds
    double              *kappa_d;                   // the hamiltonian on the GPU
    const int           blockSize = 128;            // The number of threads to launch per block

    // magma variables for ssyevr
    double      aux_work[1];            // To get optimal size of lwork
    magma_int_t aux_iwork[1], info;     // To get optimal liwork, and return info
    magma_int_t ldkappa, lwork, liwork; // Leading dim of kappa, sizes of work arrays
    magma_int_t *iwork;                 // Work array
    double      *work;                  // Work array
    double      *w   ;                  // Eigenvalues
    double      *wA  ;                  // Work array

    // magma variables for gemv
    magma_queue_t   queue;

    // variables for spectrum calculations
    double      *w_d;                   // Eigenvalues on the GPU
    double      *omega, *omega_d;       // Frequencies on CPU and GPU
    double      *Sw, *Sw_d;             // Spectral density on CPU and GPU
    double      *tmpSw;                 // Temporary spectral density

    // variables for TCF
    magmaDoubleComplex *F, *F_d, *Ftmp_d;    // F matrix on CPU and GPU
    magmaDoubleComplex *prop, *prop_d;       // Propigator matrix on CPU and GPU
    magmaDoubleComplex *ctmpmat_d;           // temporary complex matrix for matrix multiplications on gpu
    magmaDoubleComplex *ckappa_d;            // A complex version of kappa // TODO: CAN WE JUST CAST AS TYPE INSTEAD OF HAVING VARIABLES FOR THIS?
    magmaDoubleComplex tcfx, tcfy, tcfz;     // Time correlation function, polarized
    magmaDoubleComplex dcy, tcftmp;          // Decay constant and a temporary variable for the tcf
    magmaDoubleComplex *pdtcf, *pdtcf_d;     // padded time correlation functions
    magmaDoubleComplex *tcf, *tcf_d;         // Time correlation function
    magmaDoubleComplex *tmptcf;              // A temporary function for time correlation function
    double             *Ftcf, *Ftcf_d;       // Fourier transformed time correlation function
    double             *tmpFtcf;             // Temporary Fourier transformed time correlation function
    double             *time;                // Time array for tcf
    double             arg;                  // argument of exponential

    // For fft on gpu
    hipfftHandle       plan;

    // **************************************************** //
    // ***         End  Variable Declaration            *** //


    



    // ***          Begin main routine                  *** //
    // **************************************************** //

    // Open trajectory file and get info about the systeem
    printf("Will read the trajectory from: %s.\n",gmxf);
    XDRFILE *trj = xdrfile_open( gmxf, "r" ); 

    read_xtc_natoms( (char *)gmxf, &natoms);
    nmol         = natoms / natom_mol;
    nchrom       = nmol * nchrom_mol;
    ldkappa      = (magma_int_t) nchrom;

    printf("Found %d atoms and %d molecules.\n",natoms, nmol);
    printf("Found %d chromophores.\n",nchrom);


    // ***              MEMORY ALLOCATION               *** //
    // **************************************************** //

    // determine the number of blocks to launch on the gpu 
    // each thread takes care of one chromophore
    const int numBlocks = (nchrom+blockSize-1)/blockSize;
    
    // Initialize magma math library and initialize queue
    magma_init();
    magma_queue_create( 0, &queue ); 

    // allocate memory for arrays on the CPU
    x       = (rvec*)                malloc( natoms    *    sizeof(x[0] ));
    omega   = (double *)             malloc( nomega    *    sizeof(double));
    Sw      = (double *)             calloc( nomega       , sizeof(double));
    tmpSw   = (double *)             malloc( nomega    *    sizeof(double));
    time    = (double *)             malloc( ntcfpoints*    sizeof(double));
    Ftcf    = (double *)             calloc( ntcfpointsR  , sizeof(double));
    tmpFtcf = (double *)             malloc( ntcfpointsR*   sizeof(double));
    tmptcf  = (magmaDoubleComplex *) malloc( ntcfpoints*    sizeof(magmaDoubleComplex));
    tcf     = (magmaDoubleComplex *) calloc( ntcfpoints   , sizeof(magmaDoubleComplex));
    F       = (magmaDoubleComplex *) calloc( nchrom*nchrom, sizeof(magmaDoubleComplex));
    prop    = (magmaDoubleComplex *) calloc( nchrom*nchrom, sizeof(magmaDoubleComplex));


    
    // allocate memory for arrays on the GPU
    hipMalloc( &x_d     , natoms*sizeof(x[0]));
    hipMalloc( &mux_d   , nchrom*sizeof(double));
    hipMalloc( &muy_d   , nchrom*sizeof(double));
    hipMalloc( &muz_d   , nchrom*sizeof(double));
    hipMalloc( &MUX_d   , nchrom*sizeof(double));
    hipMalloc( &MUY_d   , nchrom*sizeof(double));
    hipMalloc( &MUZ_d   , nchrom*sizeof(double));
    hipMalloc( &omega_d , nomega*sizeof(double));
    hipMalloc( &Sw_d    , nomega*sizeof(double));
    hipMalloc( &Ftcf_d  , ntcfpointsR*sizeof(double));
    hipMalloc( &cmux_d  , nchrom*sizeof(magmaDoubleComplex));
    hipMalloc( &cmuy_d  , nchrom*sizeof(magmaDoubleComplex));
    hipMalloc( &cmuz_d  , nchrom*sizeof(magmaDoubleComplex));
    hipMalloc( &cmux0_d , nchrom*sizeof(magmaDoubleComplex));
    hipMalloc( &cmuy0_d , nchrom*sizeof(magmaDoubleComplex));
    hipMalloc( &cmuz0_d , nchrom*sizeof(magmaDoubleComplex));
    hipMalloc( &tmpmu_d , nchrom*sizeof(magmaDoubleComplex));

    magma_dmalloc( &eproj_d     , nchrom );
    magma_dmalloc( &kappa_d     , ldkappa*nchrom);
    magma_zmalloc( &ckappa_d    , nchrom*nchrom);
    magma_zmalloc( &F_d         , nchrom*nchrom);
    magma_zmalloc( &Ftmp_d      , nchrom*nchrom);
    magma_zmalloc( &prop_d      , nchrom*nchrom);
    magma_zmalloc( &ctmpmat_d   , nchrom*nchrom);
    magma_dmalloc( &w_d         , nchrom );
    magma_zmalloc( &tcf_d       , ntcfpoints);


    // ***          END MEMORY ALLOCATION               *** //
    // **************************************************** //
    

    // **************************************************** //
    // ***          OUTER LOOP OVER SAMPLES             *** //

    while( currentSample < nsamples )
    {
        // search trajectory for current sample starting point
        xdrinfo = read_xtc( trj, natoms, &step, &gmxtime, box, x, &prec );
        if ( xdrinfo != 0 )
        {
            printf("WARNING:: read_xtc returned error %d.\nIs the trajectory long enough?\n", xdrinfo);
            exit(0);
        }
        if ( currentSample * sampleEvery == (int) gmxtime )
        {
            printf("Now processing sample %d starting at %.2f ps\n", currentSample, gmxtime );


        // **************************************************** //
        // ***         MAIN LOOP OVER TRAJECTORY            *** //
        for ( frame = 0; frame < ntcfpoints; frame++ )
        {



            // ---------------------------------------------------- //
            // ***          Get Info About The System           *** //

            // read the current frame from the trajectory file and copy to device memory
            if ( frame != 0 ){
                // note it was read in the outer loop if we are at frame 0
                read_xtc( trj, natoms, &step, &gmxtime, box, x, &prec );
            }
            hipMemcpy( x_d, x, natoms*sizeof(x[0]), hipMemcpyHostToDevice );
            boxl = box[0][0];   // assume a square box NOTE: CHANGE IF NOT THE CASE


            // launch kernel to calculate the electric field projection along OH bonds
            get_eproj_GPU <<<numBlocks,blockSize>>> ( x_d, boxl, natoms, natom_mol, nchrom, nchrom_mol, nmol, eproj_d );

            // launch kernel to build the exciton Hamiltonian
            get_kappa_GPU <<<numBlocks,blockSize>>> ( x_d, boxl, natoms, natom_mol, nchrom, nchrom_mol, nmol, eproj_d, 
                                                      kappa_d, mux_d, muy_d, muz_d );

            // ***          Done getting System Info            *** //
            // ---------------------------------------------------- //




            // ---------------------------------------------------- //
            // ***          Diagonalize the Hamiltonian         *** //

            // if the first time, query for optimal workspace dimensions
            if ( frame == 0 && currentSample == 0)
            {
                magma_dsyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, NULL, ldkappa, 
                                  NULL, NULL, (magma_int_t) nchrom, aux_work, -1, aux_iwork, -1, &info );

                lwork   = (magma_int_t) aux_work[0];
                liwork  = aux_iwork[0];

                // make space for work arrays and eigenvalues and other stuff
                magma_dmalloc_cpu   ( &w  , (magma_int_t) nchrom );
                magma_dmalloc_pinned( &wA , (magma_int_t) nchrom*ldkappa );
                magma_dmalloc_pinned( &work , lwork  );
                magma_imalloc_cpu   ( &iwork, liwork );
            }
            magma_dsyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, kappa_d, ldkappa,
                              w, wA, ldkappa, work, lwork, iwork, liwork, &info );

            // ***          Done with the Diagonalization       *** //
            // ---------------------------------------------------- //



            // ---------------------------------------------------- //
            // ***              The Spectral Density            *** //

            if ( frame == 0 ){

                // project the transition dipole moments onto the eigenbasis
                // MU_d = kappa_d**T x mu_d 
                magma_dgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, ldkappa, mux_d, 1, 0.0, MUX_d, 1, queue);
                magma_dgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, ldkappa, muy_d, 1, 0.0, MUY_d, 1, queue);
                magma_dgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, ldkappa, muz_d, 1, 0.0, MUZ_d, 1, queue);

                // Define the spectral range of interest and initialize spectral arrays
                for (int i = 0; i < nomega; i++)
                {
                    omega[i] = (double) (omegaStart + omegaStep*i); 
                    tmpSw[i] = 0.0;
                }

                // Copy relevant variables to device memory
                hipMemcpy( omega_d, omega, nomega*sizeof(double), hipMemcpyHostToDevice );
                hipMemcpy( w_d    , w    , nchrom*sizeof(double), hipMemcpyHostToDevice );
                hipMemcpy( Sw_d   , tmpSw, nomega*sizeof(double), hipMemcpyHostToDevice );

                // calculate the spectral density on the GPU and copy back to the CPU
                get_spectral_density<<<numBlocks,blockSize>>>( w_d, MUX_d, MUY_d, MUZ_d, omega_d, Sw_d, 
                                                               nomega, nchrom, t1 );
                hipMemcpy( tmpSw, Sw_d, nomega*sizeof(double), hipMemcpyDeviceToHost );

                // Copy temporary to persistant to get average spectral density over samples
                for (int i = 0; i < nomega; i++ )
                {
                    Sw[i] += tmpSw[i];
                }
            }

            // ***           Done the Spectral Density          *** //
            // ---------------------------------------------------- //



            // ---------------------------------------------------- //

            // ***           Time Correlation Function          *** //

            // cast variables to complex to calculate time correlation function (which is complex)
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( kappa_d, ckappa_d, nchrom*nchrom );
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( mux_d  , cmux_d  , nchrom        );
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muy_d  , cmuy_d  , nchrom        );
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muz_d  , cmuz_d  , nchrom        );


            // First calculate the propigation matrix in the local basis
            if ( frame == 0 )
            {
                // initialize the F matrix at t=0 to the unit matrix
                for ( int i = 0; i < nchrom; i ++ )
                {
                    for ( int j = 0; j < nchrom; j ++ )
                    {
                        F[ i*nchrom + j] = MAGMA_Z_ZERO;
                    }
                    F[ i*nchrom + i] = MAGMA_Z_ONE;
                }
                // copy the F matrix to device memory -- after initialization, won't need back in host memory
                hipMemcpy( F_d, F, nchrom*nchrom*sizeof(magmaDoubleComplex), hipMemcpyHostToDevice );

                // set the transition dipole moment at t=0
                cast_to_complex_GPU <<<numBlocks,blockSize>>> ( mux_d  , cmux0_d  , nchrom );
                cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muy_d  , cmuy0_d  , nchrom );
                cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muz_d  , cmuz0_d  , nchrom );
            }
            else
            {
                // build the propigator
                for ( int i = 0; i < nchrom; i++ )
                {
                    // zero matrix
                    for ( int j = 0; j < nchrom; j ++ )
                    {
                        prop[ i*nchrom + j] = MAGMA_Z_ZERO;
                    }
                    // P = exp(iwt/hbar)
                    arg   = ((w[i] - avef)* dt / HBAR);
                    prop[ i*nchrom + i ] = MAGMA_Z_MAKE( cos(arg), sin(arg) );
                }

                // copy the propigator to the gpu and convert to the local basis                
                hipMemcpy( prop_d, prop, nchrom*nchrom*sizeof(magmaDoubleComplex), hipMemcpyHostToDevice );

                // ctmpmat_d = ckappa_d * prop_d
                magma_zgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             (magma_int_t) nchrom, MAGMA_Z_ONE, ckappa_d, ldkappa, prop_d, ldkappa,
                              MAGMA_Z_ZERO, ctmpmat_d, ldkappa, queue );

                // prop_d = ctmpmat_d * ckappa_d **T 
                magma_zgemm( MagmaNoTrans, MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             (magma_int_t) nchrom, MAGMA_Z_ONE, ctmpmat_d, ldkappa, ckappa_d, ldkappa, 
                             MAGMA_Z_ZERO, prop_d, ldkappa, queue );

                // propigate the F matrix in the local basis
                // ctmpmat_d = prop_d * F
                magma_zgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             (magma_int_t) nchrom, MAGMA_Z_ONE, prop_d, ldkappa, F_d, ldkappa, 
                             MAGMA_Z_ZERO, ctmpmat_d, ldkappa, queue );
                // copy the F matrix back from the temporary variable to F_d
                copy_complex_GPU <<<numBlocks,blockSize>>> ( F_d  , ctmpmat_d  , nchrom*nchrom );
            }


            // calculate mFm for x y and z components
            // tcfx = cmux0_d**T * F_d *cmux_d
            // x
            magma_zgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_Z_ONE, F_d, ldkappa,
                         cmux0_d, 1, MAGMA_Z_ZERO, tmpmu_d, 1, queue);
            tcfx = magma_zdotu( (magma_int_t) nchrom, cmux_d, 1, tmpmu_d, 1, queue );

            // y
            magma_zgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_Z_ONE, F_d, ldkappa,
                         cmuy0_d, 1, MAGMA_Z_ZERO, tmpmu_d, 1, queue);
            tcfy = magma_zdotu( (magma_int_t) nchrom, cmuy_d, 1, tmpmu_d, 1, queue );

            // z
            magma_zgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_Z_ONE, F_d, ldkappa,
                         cmuz0_d, 1, MAGMA_Z_ZERO, tmpmu_d, 1, queue);
            tcfz = magma_zdotu( (magma_int_t) nchrom, cmuz_d, 1, tmpmu_d, 1, queue );

            // save the variables to print later and multiply by the decay
            time[frame]           = dt * frame;
            tcftmp                = MAGMA_Z_ADD( tcfx  , tcfy );
            tcftmp                = MAGMA_Z_ADD( tcftmp, tcfz );
            dcy                   = MAGMA_Z_MAKE(exp( -1.0 * frame * dt / ( 2.0 * t1 )), 0.0);
            tmptcf[frame]         = MAGMA_Z_MUL( tcftmp, dcy );
 
            // ***        Done with Time Correlation            *** //
            // ---------------------------------------------------- //

        }

        // copy time correlation function to persistant memory to calculate average spectrum
        for ( int i = 0; i < ntcfpoints; i ++ )
        {
            tcf[i]  = MAGMA_Z_ADD( tcf[i] , tmptcf[i]);
        }

        // done with current sample, move to next
        currentSample +=1;
        }
    } // end outer loop


    // close xdr file
    xdrfile_close(trj);


    // pad the time correlation function with zeros, copy to device memory and perform fft
    // fourier transform the time correlation function on the GPU
    pdtcf = (magmaDoubleComplex *) calloc( ntcfpoints+nzeros, sizeof(magmaDoubleComplex));
    for ( int i = 0; i < ntcfpoints; i++ )
    {
        pdtcf[i] = tcf[i];
    }
    for ( int i = 0; i < nzeros; i++ )
    {
        pdtcf[i+ntcfpoints] = MAGMA_Z_ZERO;
    }

    magma_zmalloc( &pdtcf_d    , ntcfpoints+nzeros);
    hipMemcpy( pdtcf_d, pdtcf, (ntcfpoints+nzeros)*sizeof(magmaDoubleComplex), hipMemcpyHostToDevice );
    hipfftPlan1d( &plan, ntcfpoints+nzeros, HIPFFT_Z2D, 1);
    hipfftExecZ2D( plan, pdtcf_d, Ftcf_d );
    hipMemcpy( Ftcf, Ftcf_d, ntcfpointsR*sizeof(double), hipMemcpyDeviceToHost );
    hipfftDestroy(plan);

    // normalize spectra by number of samples
    for ( int i = 0; i < ntcfpoints; i++ )
    {
        Ftcf[i] = Ftcf[i] / (double) nsamples; 
        tcf[i]  = MAGMA_Z_DIV( tcf[i] , MAGMA_Z_MAKE( nsamples, 0.0 ));
    }
    for ( int i = 0; i < nomega; i++)
    {
        Sw[i]   = Sw[i] / (double) nsamples;
    }

    // write time correlation function
    // TODO:: allow user to define custom names
    FILE *rtcf = fopen("tcf_real.dat", "w");
    FILE *itcf = fopen("tcf_imag.dat", "w");
    for ( int i = 0; i < ntcfpoints; i++ )
    {
        fprintf( rtcf, "%g %g \n", time[i], MAGMA_Z_REAL( tcf[i] ) );
        fprintf( itcf, "%g %g \n", time[i], MAGMA_Z_IMAG( tcf[i] ) );
    }
    fclose( rtcf );
    fclose( itcf );

    // write the spectral density to file
    FILE *spec_density = fopen("spectral_density.dat", "w");
    for ( int i = 0; i < nomega; i++)
    {
        fprintf(spec_density, "%e %e\n", omega[i], Sw[i]);
    }
    fclose(spec_density);

    // Write the absorption lineshape... Since the C2R transform is inverse by default, the frequencies have to be negated
    FILE *spec_lineshape = fopen("spectral_lineshape.dat", "w");
    double factor  = 2*PI*HBAR/(dt*(ntcfpoints+nzeros));  // conversion factor to give energy and correct intensity from FFT
    for ( int i = (ntcfpoints+nzeros)/2; i < ntcfpoints+nzeros; i++ ) // "negative" FFT frequencies
    {
        if ( -1*(i-ntcfpoints-nzeros)*factor + avef <= (double) omegaStop  )
        {
            fprintf(spec_lineshape, "%e %e\n", -1*(i-ntcfpoints-nzeros)*factor + avef, Ftcf[i]/2.0);// /(factor*ntcfpoints));
        }
    }
    for ( int i = 0; i < ntcfpoints+nzeros / 2 ; i++)       // "positive" FFT frequencies
    {
        if ( -1*i*factor + avef >= (double) omegaStart)
        {
            fprintf(spec_lineshape, "%e %e\n", -1*i*factor + avef, Ftcf[i]/2.0);// /(factor*ntcfpoints));
        }
    }
    fclose(spec_lineshape);

    // free memory on the CPU and GPU and finalize magma library
    magma_queue_destroy( queue );

    free(x);
    free(omega);
    free(Sw);
    free(tmpSw);
    free(time);
    free(Ftcf);
    free(tmpFtcf);
    free(tcf);
    free(F);
    free(prop);
    free(pdtcf);

    hipFree(x_d);
    hipFree(mux_d); 
    hipFree(muy_d);
    hipFree(muz_d);
    hipFree(MUX_d); 
    hipFree(MUY_d);
    hipFree(MUZ_d);
    hipFree(omega_d);
    hipFree(Sw_d);
    hipFree(Ftcf_d);
    hipFree(cmux_d); 
    hipFree(cmuy_d);
    hipFree(cmuz_d);
    hipFree(cmux0_d); 
    hipFree(cmuy0_d);
    hipFree(cmuz0_d);
    hipFree(tmpmu_d);
 
    magma_free(eproj_d);
    magma_free(kappa_d);
    magma_free(ckappa_d);
    magma_free(F_d);
    magma_free(Ftmp_d);
    magma_free(prop_d);
    magma_free(ctmpmat_d);
    magma_free(w_d);
    magma_free(tcf_d);
    magma_free(pdtcf_d);

    magma_free_cpu(w);
    magma_free_cpu(iwork);
    magma_free_pinned( work );
    magma_free_pinned( wA );

    // final call to finalize magma math library
    magma_finalize();

    return 0;
}

/**********************************************************
   
   BUILD ELECTRIC FIELD PROJECTION ALONG OH BONDS
                    GPU FUNCTION

 **********************************************************/
__global__
void get_eproj_GPU( rvec *x, float boxl, int natoms, int natom_mol, int nchrom, int nchrom_mol, int nmol, double  *eproj )
{
    
    int n, m, i, j, istart, istride;
    int chrom;
    double mox[DIM];                     // oxygen position on molecule m
    double mx[DIM];                      // atom position on molecule m
    double nhx[DIM];                     // hydrogen position on molecule n of the current chromophore
    double nox[DIM];                     // oxygen position on molecule n
    double nohx[DIM];                    // the unit vector pointing along the OH bond for the current chromophore
    double dr[DIM];                      // the min image vector between two atoms
    double r;                            // the distance between two atoms 
    const float cutoff = 0.7831;         // the oh cutoff distance
    const float bohr_nm = 18.8973;       // convert from bohr to nanometer
    double efield[DIM];                  // the electric field vector

    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // Loop over the chromophores belonging to the current thread
    for ( chrom = istart; chrom < nchrom; chrom += istride )
    {
        // calculate the molecule hosting the current chromophore 
        n = chrom / nchrom_mol;

        // initialize the electric field vector to zero at this chromophore
        efield[0]   =   0.;
        efield[1]   =   0.;
        efield[2]   =   0.;


        // ***          GET INFO ABOUT MOLECULE N HOSTING CHROMOPHORE       *** //
        //                      N IS OUR REFERENCE MOLECULE                     //
        // get the position of the hydrogen associated with the current stretch 
        // NOTE: I'm making some assumptions about the ordering of the positions, 
        // this can be changed if necessary for a more robust program
        // Throughout, I assume that the atoms are grouped into molecules and that
        // every 4th molecule starting at 0 (1, 2, 3) is OW (HW1, HW2, MW)
        if ( chrom % 2 == 0 ){      //HW1
            nhx[0]  = x[ n*natom_mol + 1 ][0];
            nhx[1]  = x[ n*natom_mol + 1 ][1];
            nhx[2]  = x[ n*natom_mol + 1 ][2];
        }
        else if ( chrom % 2 == 1 ){ //HW2
            nhx[0]  = x[ n*natom_mol + 2 ][0];
            nhx[1]  = x[ n*natom_mol + 2 ][1];
            nhx[2]  = x[ n*natom_mol + 2 ][2];
        }

        // The oxygen position
        nox[0]  = x[ n*natom_mol ][0];
        nox[1]  = x[ n*natom_mol ][1];
        nox[2]  = x[ n*natom_mol ][2];

        // The oh unit vector
        nohx[0] = minImage( nhx[0] - nox[0], boxl );
        nohx[1] = minImage( nhx[1] - nox[1], boxl );
        nohx[2] = minImage( nhx[2] - nox[2], boxl );
        r       = mag3(nohx);
        nohx[0] /= r;
        nohx[1] /= r;
        nohx[2] /= r;
        // for testing with YICUN -- can change to ROH later...
        //nohx[0] /= 0.09572;
        //nohx[1] /= 0.09572;
        //nohx[2] /= 0.09572;
 
        // ***          DONE WITH MOLECULE N                                *** //



        // ***          LOOP OVER ALL OTHER MOLECULES                       *** //
        for ( m = 0; m < nmol; m++ ){

            // skip the reference molecule
            if ( m == n ) continue;

            // get oxygen position on current molecule
            mox[0] = x[ m*natom_mol ][0];
            mox[1] = x[ m*natom_mol ][1];
            mox[2] = x[ m*natom_mol ][2];

            // find displacement between oxygen on m and hydrogen on n
            dr[0]  = minImage( mox[0] - nhx[0], boxl );
            dr[1]  = minImage( mox[1] - nhx[1], boxl );
            dr[2]  = minImage( mox[2] - nhx[2], boxl );
            r      = mag3(dr);

            // skip if the distance is greater than the cutoff
            if ( r > cutoff ) continue;

            // loop over all atoms in the current molecule and calculate the electric field 
            // (excluding the oxygen atoms since they have no charge)
            for ( i=1; i < natom_mol; i++ ){

                // position of current atom
                mx[0] = x[ m*natom_mol + i ][0];
                mx[1] = x[ m*natom_mol + i ][1];
                mx[2] = x[ m*natom_mol + i ][2];

                // the minimum image displacement between the reference hydrogen and the current atom
                // NOTE: this converted to bohr so the efield will be in au
                dr[0]  = minImage( nhx[0] - mx[0], boxl )*bohr_nm;
                dr[1]  = minImage( nhx[1] - mx[1], boxl )*bohr_nm;
                dr[2]  = minImage( nhx[2] - mx[2], boxl )*bohr_nm;
                r      = mag3(dr);

                // Add the contribution of the current atom to the electric field
                if ( i < 3  ){              // HW1 and HW2
                    for ( j=0; j < DIM; j++){
                        efield[j] += 0.52 * dr[j] / (r*r*r);
                    }
                }
                else if ( i == 3 ){         // MW (note the negative sign)
                    for ( j=0; j < DIM; j++){
                        efield[j] -= 1.04 * dr[j] / (r*r*r);
                    }
                }
            } // end loop over atoms in molecule m

        } // end loop over molecules m

        // project the efield along the OH bond to get the relevant value for the map
        eproj[chrom] = dot3( efield, nohx );

        // test looks good, everything appears to be ok -- a little different than YICUN, but i think it is numerical error
        /*
        if( chrom == 0 ){
            printf("chrom %d En %g\n", chrom, eproj[chrom]);
            printf("%g %g %g\n", efield[0], efield[1], efield[2]);
            printf("%g %g %g\n", nohx[0], nohx[1], nohx[2]);
        }
        */


        // printf("chrom: %d, eproj %f \n", chrom, eproj[chrom]);

    } // end loop over reference chromophores

}

/**********************************************************
   
   BUILD HAMILTONIAN AND RETURN TRANSITION DIPOLE VECTOR
    FOR EACH CHROMOPHORE ON THE GPU

 **********************************************************/
__global__
void get_kappa_GPU( rvec *x, float boxl, int natoms, int natom_mol, int nchrom, int nchrom_mol, int nmol, 
                    double *eproj, double *kappa, double *mux, double *muy, double *muz)
{
    
    int n, m, istart, istride;
    int chromn, chromm;
    double mox[DIM];                         // oxygen position on molecule m
    double mhx[DIM];                         // atom position on molecule m
    double nhx[DIM];                         // hydrogen position on molecule n of the current chromophore
    double nox[DIM];                         // oxygen position on molecule n
    double noh[DIM];
    double moh[DIM];
    double nmu[DIM];
    double mmu[DIM];
    double mmuprime;
    double nmuprime;
    double dr[DIM];                          // the min image vector between two atoms
    double r;                                // the distance between two atoms 
    const double bohr_nm    = 18.8973;       // convert from bohr to nanometer
    const double cm_hartree = 2.1947463E5;   // convert from cm-1 to hartree
    double En, Em;                           // the electric field projection
    double xn, xm, pn, pm;                   // the x and p from the map
    double wn, wm;                           // the energies

    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // Loop over the chromophores belonging to the current thread and fill in kappa for that row
    for ( chromn = istart; chromn < nchrom; chromn += istride )
    {
        // calculate the molecule hosting the current chromophore 
        // and get the corresponding electric field at the relevant hydrogen
        n   = chromn / nchrom_mol;
        En  = eproj[chromn];

        // build the map
        wn  = 3760.2 - 3541.7*En - 152677.0*En*En;
        xn  = 0.19285 - 1.7261E-5 * wn;
        pn  = 1.6466  + 5.7692E-4 * wn;
        nmuprime = 0.1646 + 11.39*En + 63.41*En*En;

        // and calculate the location of the transition dipole moment
        // SEE calc_efield_GPU for assumptions about ordering of atoms
        nox[0]  = x[ n*natom_mol ][0];
        nox[1]  = x[ n*natom_mol ][1];
        nox[2]  = x[ n*natom_mol ][2];
        if ( chromn % 2 == 0 )       //HW1
        {
            nhx[0]  = x[ n*natom_mol + 1 ][0];
            nhx[1]  = x[ n*natom_mol + 1 ][1];
            nhx[2]  = x[ n*natom_mol + 1 ][2];
        }
        else if ( chromn % 2 == 1 )  //HW2
        {
            nhx[0]  = x[ n*natom_mol + 2 ][0];
            nhx[1]  = x[ n*natom_mol + 2 ][1];
            nhx[2]  = x[ n*natom_mol + 2 ][2];
        }

        // The OH unit vector
        noh[0] = minImage( nhx[0] - nox[0], boxl );
        noh[1] = minImage( nhx[1] - nox[1], boxl );
        noh[2] = minImage( nhx[2] - nox[2], boxl );
        r      = mag3(noh);
        noh[0] /= r;
        noh[1] /= r;
        noh[2] /= r;

        // The location of the TDM
        nmu[0] = minImage( nox[0] + 0.067 * noh[0], boxl );
        nmu[1] = minImage( nox[1] + 0.067 * noh[1], boxl );
        nmu[2] = minImage( nox[2] + 0.067 * noh[2], boxl );
        
        // and the TDM vector to return
        mux[chromn] = noh[0] * nmuprime * xn;
        muy[chromn] = noh[1] * nmuprime * xn;
        muz[chromn] = noh[2] * nmuprime * xn;



        // Loop over all other chromophores
        for ( chromm = 0; chromm < nchrom; chromm ++ )
        {
            // calculate the molecule hosting the current chromophore 
            // and get the corresponding electric field at the relevant hydrogen
            m   = chromm / nchrom_mol;
            Em  = eproj[chromm];

            // also get the relevent x and p from the map
            wm  = 3760.2 - 3541.7*Em - 152677.0*Em*Em;
            xm  = 0.19285 - 1.7261E-5 * wm;
            pm  = 1.6466  + 5.7692E-4 * wm;
            mmuprime = 0.1646 + 11.39*Em + 63.41*Em*Em;

            // the diagonal energy
            if ( chromn == chromm )
            {
                //kappa[chromn*nchrom + chromm]   =   3500.0;
                //continue;
                // Note that this is a flattened 2d array
                kappa[chromn*nchrom + chromm]   =   wm; 
            }

            // intramolecular coupling
            else if ( m == n )
            {
                //kappa[chromn*nchrom + chromm]   =   0.0;
                //continue;
                kappa[chromn*nchrom + chromm]   =  (-1361.0 + 27165*(En + Em))*xn*xm - 1.887*pn*pm;
            }

            // intermolecular coupling
            else
            {
                
                //kappa[chromn*nchrom + chromm]   =   0.0;
                //continue;
                
                // calculate the distance between dipoles
                // they are located 0.67 A from the oxygen along the OH bond
                // tdm position on chromophore n
                mox[0]  = x[ m*natom_mol ][0];
                mox[1]  = x[ m*natom_mol ][1];
                mox[2]  = x[ m*natom_mol ][2];
                if ( chromm % 2 == 0 )       //HW1
                {
                    mhx[0]  = x[ m*natom_mol + 1 ][0];
                    mhx[1]  = x[ m*natom_mol + 1 ][1];
                    mhx[2]  = x[ m*natom_mol + 1 ][2];
                }
                else if ( chromm % 2 == 1 )  //HW2
                {
                    mhx[0]  = x[ m*natom_mol + 2 ][0];
                    mhx[1]  = x[ m*natom_mol + 2 ][1];
                    mhx[2]  = x[ m*natom_mol + 2 ][2];
                }

                // The OH unit vector
                moh[0] = minImage( mhx[0] - mox[0], boxl );
                moh[1] = minImage( mhx[1] - mox[1], boxl );
                moh[2] = minImage( mhx[2] - mox[2], boxl );
                r      = mag3(moh);
                moh[0] /= r;
                moh[1] /= r;
                moh[2] /= r;

                // The location of the TDM and the dipole derivative
                mmu[0] = minImage( mox[0] + 0.067 * moh[0], boxl );
                mmu[1] = minImage( mox[1] + 0.067 * moh[1], boxl );
                mmu[2] = minImage( mox[2] + 0.067 * moh[2], boxl );

                // the distance between TDM on N and on M and convert to unit vector
                dr[0] = minImage( nmu[0] - mmu[0], boxl );
                dr[1] = minImage( nmu[1] - mmu[1], boxl );
                dr[2] = minImage( nmu[2] - mmu[2], boxl );
                r     = mag3( dr );
                dr[0] /= r;
                dr[1] /= r;
                dr[2] /= r;
                r     *= bohr_nm; // convert to bohr

                // The coupling in the transition dipole approximation in wavenumber
                // Note the conversion to wavenumber
                kappa[chromn*nchrom + chromm]   = ( dot3( noh, moh ) - 3.0 * dot3( noh, dr ) * 
                                                    dot3( moh, dr ) ) / ( r*r*r ) * 
                                                    xn*xm*nmuprime*mmuprime*cm_hartree;
            }// end intramolecular coupling
        }// end loop over chromm
    }// end loop over reference
}


/**********************************************************
   
        Calculate the Spectral Density

 **********************************************************/
__global__
void get_spectral_density( double *w, double *MUX, double *MUY, double *MUZ, double *omega, double *Sw, 
                           int nomega, int nchrom, double t1 ){

    int istart, istride, i, chromn;
    double wi, dw, MU2, gamma;
    
    // split up each desired frequency to separate thread on GPU
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // the linewidth parameter
    gamma = HBAR/(t1 * 2.0);

    // Loop over the chromophores belonging to the current thread and fill in kappa for that row
    for ( i = istart; i < nomega; i += istride )
    {
        // get current frequency
        wi = omega[i];
        
        // Loop over all chromophores calculatint the spectral intensity at the current frequency
        for ( chromn = 0; chromn < nchrom; chromn ++ ){
            // calculate the TDM squared and get the mode energy
            MU2     = MUX[chromn]*MUX[chromn] + MUY[chromn]*MUY[chromn] + MUZ[chromn]*MUZ[chromn];
            dw      = wi - w[chromn];

            // add a lorentzian lineshape to the spectral density
            Sw[i] += MU2 * gamma / ( dw*dw + gamma*gamma )/PI;
        }
        //printf("Sw[%d]=%f\n", i, Sw[i]);
    }
}

/**********************************************************
   
        HELPER FUNCTIONS FOR GPU CALCULATIONS
            CALLABLE FROM CPU AND GPU

 **********************************************************/

// The minimage image of a scalar
double minImage( double dx, double boxl )
{
    return dx - boxl*round(dx/boxl);
}

// The magnitude of a 3 dimensional vector
double mag3( double dx[3] )
{
    return sqrt( dot3( dx, dx ) );
}

// The dot product of a 3 dimensional vector
double dot3( double x[3], double y[3] )
{
    return  x[0]*y[0] + x[1]*y[1] + x[2]*y[2];
}

// cast the matrix from float to complex -- this may not be the best way to do this, but it is quick to implement
__global__
void cast_to_complex_GPU ( double *d_d, magmaDoubleComplex *z_d, int n )
{
    int istart, istride, i;
    
    // split up each desired frequency to separate thread on GPU
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // convert from float to complex
    for ( i = istart; i < n; i += istride )
    {
        z_d[i] = MAGMA_Z_MAKE( d_d[i], 0.0 ); 
    }
}

__global__
void copy_complex_GPU( magmaDoubleComplex *out_d, magmaDoubleComplex *in_d, int n )
{
    int istart, istride, i;
    
    // split up each desired frequency to separate thread on GPU
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // convert from float to complex
    for ( i = istart; i < n; i += istride )
    {
        out_d[i] = in_d[i];
    }
}
