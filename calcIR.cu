#include "hip/hip_runtime.h"
/*This is my first attempt to port my python ir program to cuda. 
 * It currently suffers from **very** slow excecution in python. 
 * I'm going to try to port it to cuda c */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <xdrfile/xdrfile.h>
#include <xdrfile/xdrfile_xtc.h>
#include "calcIR.h" 
#include <complex.h>
#include <time.h>
#include <unistd.h>
#include <signal.h>

#include "magma_v2.h"
#include <hipfft/hipfft.h>

// Global variable to catch signals and write checkpoint file
volatile sig_atomic_t interrupted=false;

int main(int argc, char *argv[])
{

    // Some help for starting the program
    if ( argc != 2 ){
        printf("Usage:\n\tInclude as the first argument either the name of an input file,  or a checkpoint\n\tfile with extension '.cpt' if restarting the calculation. No other arguments are\n\tallowed.\n");
        exit(EXIT_FAILURE);   
    }
    

    // register signal handler
    signal( SIGINT, signal_handler );
    signal( SIGTERM, signal_handler );


    // ***              Variable Declaration            *** //
    // **************************************************** //

    printf("\n>>> Setting default parameters\n");

    // Default values for user input
    char          gmxf[MAX_STR_LEN]; 
    strncpy( gmxf, "n216/traj_comp.xtc", MAX_STR_LEN );                   // trajectory file
    char          outf[MAX_STR_LEN]; 
    strncpy( outf, " ", MAX_STR_LEN );                                    // name for output files
    char          cptf[MAX_STR_LEN]; 
    strncpy( cptf, " ", MAX_STR_LEN );                                    // name for output files
    char          model[MAX_STR_LEN];
    strncpy( model, "tip4p", MAX_STR_LEN );
    int           imodel        = 0;
    int           SPECD_FLAG    = 1;                                      // calculate the spectral density
    int           ifintmeth     = 0;
    user_real_t   dt            = 0.010;                                  // dt between frames in xtc file (in ps)
    int           ntcfpoints    = 150 ;                                   // the number of tcf points for each spectrum
    int           nsamples      = 1   ;                                   // number of samples to average for the total spectrum
    int           sampleEvery   = 10  ;                                   // sample a new configuration every sampleEvery ps. Note ntcfpoints*dt must be less than sampleEvery.
    user_real_t   beginTime     = 0   ;                                   // the beginning time in ps to allow for equilibration

    user_real_t   t1            = 0.260;                                  // relaxation time ( in ps )
    user_real_t   avef          = 3415.2;                                 // the approximate average stretch frequency to get rid of high frequency oscillations in the time correlation function
    int           omegaStart    = 2000;                                   // starting frequency for spectral density
    int           omegaStop     = 5000;                                   // ending frequency for spectral density
    int           omegaStep     = 5;                                      // resolution for spectral density

    int           natom_mol     = 4;                                      // Atoms per water molecule  :: MODEL DEPENDENT
    int           nchrom_mol    = 2;                                      // Chromophores per molecule :: TWO for stretch -- ONE for bend
    int           nzeros        = 25600;                                  // zeros for padding fft

    user_real_t   max_int_steps = 2.0;                                    // number of Adams integration steps between each dt

 
    // get user input parameters
    if ( strstr(argv[1], ".cpt") == NULL )
    {
        // START FROM INPUT FILE
        ir_init( argv, gmxf, cptf, outf, model, &ifintmeth, &dt, &ntcfpoints, &nsamples, &sampleEvery, &t1, 
                &avef, &omegaStart, &omegaStop, &omegaStep, &natom_mol, &nchrom_mol, &nzeros, &beginTime,
                &SPECD_FLAG, &max_int_steps);
    }
    else
    {
        // START FROM CHECKPOINT FILE
        checkpoint( argv, gmxf, cptf, outf, model, &ifintmeth, &dt, &ntcfpoints, &nsamples, &sampleEvery, &t1, 
                    &avef, &omegaStart, &omegaStop, &omegaStep, &natom_mol, &nchrom_mol, &nzeros, &beginTime,
                    &SPECD_FLAG, &max_int_steps, 0, 0, NULL, NULL, NULL, NULL, NULL, NULL, NULL, NULL, CP_INIT );

        // TODO: nsamples and t1 can be changed from the values in the checkpoint file...is there an easy way to do this?
        //printf("Enter the number of samples: ");
        //scanf("%d", &nsamples);
    }

    // Print the parameters to stdout
    printf("\tSetting xtc file %s\n",                       gmxf        );
    printf("\tSetting default file name to %s\n",           outf        );
    printf("\tSetting cpt file %s\n",                       cptf        );
    printf("\tSetting model to %s\n",                       model       );
    printf("\tSetting F integration method to %d\n",        ifintmeth   );
    printf("\tSetting the number of tcf points to %d\n",    ntcfpoints  );
    printf("\tSetting nsamples to %d\n",                    nsamples    ); 
    printf("\tSetting sampleEvery to %d (ps)\n",            sampleEvery );
    printf("\tSetting omegaStart to %d\n",                  omegaStart  );
    printf("\tSetting omegaStop to %d\n",                   omegaStop   );
    printf("\tSetting omegaStep to %d\n",                   omegaStep   );
    printf("\tSetting natom_mol to %d\n",                   natom_mol   );
    printf("\tSetting nchrom_mol to %d\n",                  nchrom_mol  );
    printf("\tSetting nzeros to %d\n",                      nzeros      );
    printf("\tSetting SPECD_FLAG to %d\n",                  SPECD_FLAG  );
#ifdef USE_DOUBLES
    printf("\tSetting dt to %lf\n",                         dt          );
    printf("\tSetting t1 to %lf (ps)\n",                    t1          );
    printf("\tSetting avef to %lf\n",                       avef        );
    printf("\tSetting equilibration time to %lf (ps)\n",    beginTime   );
    printf("\tSetting max_int_steps to %lf\n",              max_int_steps );
#else
    printf("\tSetting dt to %f\n",                          dt          );
    printf("\tSetting t1 to %f (ps)\n",                     t1          );
    printf("\tSetting avef to %f\n",                        avef        );
    printf("\tSetting equilibration time to %f (ps)\n",     beginTime   );
    printf("\tSetting max_int_steps to %f\n",               max_int_steps );
#endif
 

    // Some variables and constants
    int                 natoms, nmol, nchrom;
    magma_int_t         nchrom2;
    const int           ntcfpointsR     = ( nzeros + ntcfpoints - 1 ) * 2;              // number of points for the real fourier transform
    const int           nomega          = ( omegaStop - omegaStart ) / omegaStep + 1;   // number of frequencies for the spectral density
    int                 currentSample   = 0;                                            // current sample
    int                 currentFrame    = 0;                                            // current frame

    // set model to integer to pass to gpu kernel to test in if statement for remaking OM bond lengths
    if ( strcmp( model, "tip4p2005" ) == 0 || strcmp( model, "e3b3" ) == 0 ) imodel = 1;
    else imodel = 0;

    // Trajectory stuff for the CPU
    rvec                *x;                                                             // Position vector
    matrix              box;                                                            // Box vectors
    float               boxl, gmxtime, prec;                                            // Box lengths, time at current frame, precision of xtf file
    int                 step, xdrinfo;                                                  // The current step number

    // Some variables for the GPU
    rvec                *x_d;                                                           // positions
    user_real_t         *mux_d, *muy_d, *muz_d;                                         // transition dipole moments
    user_complex_t      *cmux0_d, *cmuy0_d, *cmuz0_d;                                   // complex version of the transition dipole moment at t=0 
    user_complex_t      *cmux_d, *cmuy_d, *cmuz_d;                                      // complex versions of the transition dipole moment
    user_complex_t      *tmpmu_d;                                                       // to sum all polarizations
    user_real_t         *MUX_d, *MUY_d, *MUZ_d;                                         // transition dipole moments in the eigen basis
    user_real_t         *eproj_d;                                                       // the electric field projected along the oh bonds
    user_real_t         *kappa_d;                                                       // the hamiltonian on the GPU
    const int           blockSize = 128;                                                // The number of threads to launch per block


    // Variables for F matrix integration
    user_complex_t      *k1_d, *k2_d, *k3_d, *k4_d;                                     // Adams integration variables
    int                 order_counter = 0 ;                                             // To keep track of the current order of the Adams method

    // magma variables for ssyevr
    user_real_t         aux_work[1];                                                    // To get optimal size of lwork
    magma_int_t         aux_iwork[1], info;                                             // To get optimal liwork, and return info
    magma_int_t         lwork, liwork;                                         // Leading dim of kappa, sizes of work arrays
    magma_int_t         *iwork;                                                         // Work array
    user_real_t         *work;                                                          // Work array
    user_real_t         *w   ;                                                          // Eigenvalues
    user_real_t         *wA  ;                                                          // Work array
    int                 SSYEVD_ALLOC_FLAG = 1;                                          // flag whether to allocate ssyevr arrays -- it is turned off after they are allocated

    // magma variables for gemv
    magma_queue_t       queue;

    // variables for spectrum calculations
    user_real_t         *w_d;                                                           // Eigenvalues on the GPU
    user_real_t         *omega, *omega_d;                                               // Frequencies on CPU and GPU
    user_real_t         *Sw, *Sw_d;                                                     // Spectral density on CPU and GPU
    user_real_t         *tmpSw;                                                         // Temporary spectral density

    // variables for TCF
    user_complex_t      *F_d;                                                           // F matrix on GPU
    user_complex_t      *prop_d;                                                        // Propigator matrix on GPU
    user_complex_t      *ctmpmat_d;                                                     // temporary complex matrix for matrix multiplications on gpu
    user_complex_t      *ckappa_d;                                                      // A complex version of kappa
    user_complex_t      tcfx, tcfy, tcfz;                                               // Time correlation function, polarized
    user_complex_t      dcy, tcftmp;                                                    // Decay constant and a temporary variable for the tcf
    user_complex_t      *pdtcf, *pdtcf_d;                                               // padded time correlation functions
    user_complex_t      *tcf, *tcf_d;                                                   // Time correlation function
    user_real_t         *Ftcf, *Ftcf_d;                                                 // Fourier transformed time correlation function

    // For fft on gpu
    hipfftHandle         plan;

    // for timing
    time_t              start=time(NULL), end;

    // for file output
    FILE *rtcf;
    FILE *itcf;
    FILE *spec_density;
    FILE *spec_lineshape; 
    user_real_t factor;                                                                 // conversion factor to give energy and correct intensity from FFT
    

    // **************************************************** //
    // ***         End  Variable Declaration            *** //


    



    // ***          Begin main routine                  *** //
    // **************************************************** //

    // Open trajectory file and get info about the systeem

    XDRFILE *trj = xdrfile_open( gmxf, "r" ); 
    if ( trj == NULL )
    {
        printf("WARNING: The file %s could not be opened. Is the name correct?\n", gmxf);
        exit(EXIT_FAILURE);
    }
    printf(">>> Will read the trajectory from: %s.\n",gmxf);


    read_xtc_natoms( (char *)gmxf, &natoms);
    nmol         = natoms / natom_mol;
    nchrom       = nmol * nchrom_mol;
    nchrom2      = (magma_int_t) nchrom*nchrom;

    printf(">>> Found %d atoms and %d molecules.\n",natoms, nmol);
    printf(">>> Found %d chromophores.\n",nchrom);


    // ***              MEMORY ALLOCATION               *** //
    // **************************************************** //

    // determine the number of blocks to launch on the gpu 
    // each thread takes care of one chromophore for building the electric field and Hamiltonian
    const int numBlocks = (nchrom+blockSize-1)/blockSize;
    
    // Initialize magma math library and initialize queue
    magma_init();
    magma_queue_create( 0, &queue ); 

    // CPU arrays
    x       = (rvec*)            malloc( natoms       * sizeof(x[0] )); if ( x == NULL ) MALLOC_ERR;
    Ftcf    = (user_real_t *)    calloc( ntcfpointsR  , sizeof(user_real_t)); if ( Ftcf == NULL ) MALLOC_ERR;
    tcf     = (user_complex_t *) calloc( ntcfpoints   , sizeof(user_complex_t)); if ( tcf == NULL ) MALLOC_ERR;

    // GPU arrays
    hipError_t Cuerr;
    Cuerr = hipMalloc( &x_d      , natoms       *sizeof(x[0])); CHK_ERR;
    Cuerr = hipMalloc( &Ftcf_d   , ntcfpointsR  *sizeof(user_real_t)); CHK_ERR;
    Cuerr = hipMalloc( &mux_d    , nchrom       *sizeof(user_real_t)); CHK_ERR;
    Cuerr = hipMalloc( &muy_d    , nchrom       *sizeof(user_real_t)); CHK_ERR;
    Cuerr = hipMalloc( &muz_d    , nchrom       *sizeof(user_real_t)); CHK_ERR;
    Cuerr = hipMalloc( &eproj_d  , nchrom       *sizeof(user_real_t)); CHK_ERR;
    Cuerr = hipMalloc( &kappa_d  , nchrom2      *sizeof(user_real_t)); CHK_ERR;
    Cuerr = hipMalloc( &cmux_d   , nchrom       *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &cmuy_d   , nchrom       *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &cmuz_d   , nchrom       *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &cmux0_d  , nchrom       *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &cmuy0_d  , nchrom       *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &cmuz0_d  , nchrom       *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &tmpmu_d  , nchrom       *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &ckappa_d , nchrom2      *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &F_d      , nchrom2      *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &ctmpmat_d, nchrom2      *sizeof(user_complex_t)); CHK_ERR;
    Cuerr = hipMalloc( &tcf_d    , ntcfpoints   *sizeof(user_complex_t)); CHK_ERR;


    // memory for spectral density calculation, if requested
    if ( SPECD_FLAG )
    {
        // CPU arrays
        omega   = (user_real_t *)    malloc( nomega       * sizeof(user_real_t)); if ( omega == NULL ) MALLOC_ERR;
        Sw      = (user_real_t *)    calloc( nomega       , sizeof(user_real_t)); if ( Ftcf  == NULL ) MALLOC_ERR;
        tmpSw   = (user_real_t *)    malloc( nomega       * sizeof(user_real_t)); if ( tmpSw == NULL ) MALLOC_ERR;

        // GPU arrays
        Cuerr = hipMalloc( &MUX_d   , nchrom       *sizeof(user_real_t)); CHK_ERR;
        Cuerr = hipMalloc( &MUY_d   , nchrom       *sizeof(user_real_t)); CHK_ERR;
        Cuerr = hipMalloc( &MUZ_d   , nchrom       *sizeof(user_real_t)); CHK_ERR;
        Cuerr = hipMalloc( &omega_d , nomega       *sizeof(user_real_t)); CHK_ERR;
        Cuerr = hipMalloc( &Sw_d    , nomega       *sizeof(user_real_t)); CHK_ERR;
        Cuerr = hipMalloc( &w_d     , nchrom       *sizeof(user_real_t)); CHK_ERR;

        // initialize omega array
        for (int i = 0; i < nomega; i++) omega[i] = (user_real_t) (omegaStart + omegaStep*i); 
    }
 

    // memory for integration of F depending on which method is used
    if ( ifintmeth == 0 ) // exact
    {
        Cuerr = hipMalloc( &prop_d  , nchrom2      *sizeof(user_complex_t)); CHK_ERR;
    }
    else if ( ifintmeth == 1 ) // adams integration
    {
        Cuerr = hipMalloc( &k1_d    , nchrom2      *sizeof(user_complex_t)); CHK_ERR;
        Cuerr = hipMalloc( &k2_d    , nchrom2      *sizeof(user_complex_t)); CHK_ERR;
        Cuerr = hipMalloc( &k3_d    , nchrom2      *sizeof(user_complex_t)); CHK_ERR;
        Cuerr = hipMalloc( &k4_d    , nchrom2      *sizeof(user_complex_t)); CHK_ERR;
    }

    // ***            END MEMORY ALLOCATION             *** //
    // **************************************************** //
    

    // ***       Read State Info From Checkpoint        *** //
    // **************************************************** //
    if ( strstr(argv[1], ".cpt") != NULL )
    {
        checkpoint( argv, gmxf, cptf, outf, model, &ifintmeth, &dt, &ntcfpoints, &nsamples, &sampleEvery, &t1, 
                    &avef, &omegaStart, &omegaStop, &omegaStep, &natom_mol, &nchrom_mol, &nzeros, &beginTime,
                    &SPECD_FLAG, &max_int_steps, nchrom, nomega, &currentSample, &currentFrame, tcf, Sw, F_d, 
                    cmux0_d, cmuy0_d, cmuz0_d, CP_READ );
    }
    // **************************************************** //



    
    // **************************************************** //
    // ***          OUTER LOOP OVER SAMPLES             *** //

    printf("\n>>> Now calculate the absorption spectrum\n");
    printf("----------------------------------------------------------\n");


    while( currentSample < nsamples )
    {
        // search trajectory for current sample starting point
        xdrinfo = read_xtc( trj, natoms, &step, &gmxtime, box, x, &prec );

        if ( xdrinfo != 0 )
        {
            printf("WARNING:: read_xtc returned error %d.\nIs the trajectory long enough?\n", xdrinfo);
            exit(0);
        }

        if ( currentSample * sampleEvery + (int) beginTime == (int) gmxtime )
        {
            printf("\n    Now processing sample %d/%d starting at %.2f ps\n", currentSample + 1, nsamples, gmxtime );
            fflush(stdout);

            // If starting from checkpoint, fast forward the trajectory until you are at the correct frame 
            if ( currentFrame != 0 ) for ( int i = 0; i < currentFrame -1 ; i++ ) read_xtc( trj, natoms, &step, &gmxtime, box, x, &prec );


        // **************************************************** //
        // ***         MAIN LOOP OVER TRAJECTORY            *** //
        while( currentFrame < ntcfpoints )
        {
 
            // If the program has recieved a signal, write the current state and exit
            if ( interrupted )
            {
                checkpoint( argv, gmxf, cptf, outf, model, &ifintmeth, &dt, &ntcfpoints, &nsamples, &sampleEvery, &t1, 
                            &avef, &omegaStart, &omegaStop, &omegaStep, &natom_mol, &nchrom_mol, &nzeros, &beginTime,
                            &SPECD_FLAG, &max_int_steps, nchrom, nomega, &currentSample, &currentFrame, tcf, Sw, F_d, 
                            cmux0_d, cmuy0_d, cmuz0_d, CP_WRITE );
                exit(EXIT_SUCCESS);
            }



            // ---------------------------------------------------- //
            // ***          Get Info About The System           *** //


            // read the current frame from the trajectory file and copy to device memory
            // note it was read in the outer loop if we are at frame 0
            // also assume a square box, but this will need to be changed if it is not the case
            if ( currentFrame != 0 ){
                read_xtc( trj, natoms, &step, &gmxtime, box, x, &prec );
            }
            hipMemcpy( x_d, x, natoms*sizeof(x[0]), hipMemcpyHostToDevice );
            boxl = box[0][0];

            // launch kernel to calculate the electric field projection along OH bonds and build the exciton hamiltonian
            get_eproj_GPU <<<numBlocks,blockSize>>> ( x_d, boxl, natoms, natom_mol, nchrom, nchrom_mol, nmol, imodel, eproj_d );
            get_kappa_GPU <<<numBlocks,blockSize>>> ( x_d, boxl, natoms, natom_mol, nchrom, nchrom_mol, nmol, eproj_d, kappa_d, mux_d, muy_d, muz_d, avef );


            // ***          Done getting System Info            *** //
            // ---------------------------------------------------- //




            // ---------------------------------------------------- //
            // ***          Diagonalize the Hamiltonian         *** //

            // Note that kappa only needs to be diagonalized if the exact integration method is requested or the spectral density
            if ( ifintmeth == 0 || SPECD_FLAG )
            {

                // if the first time, query for optimal workspace dimensions
                if ( SSYEVD_ALLOC_FLAG )
                {
#ifdef USE_DOUBLES
                    magma_dsyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, NULL, (magma_int_t) nchrom, 
                                      NULL, NULL, (magma_int_t) nchrom, aux_work, -1, aux_iwork, -1, &info );
#else
                    magma_ssyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, NULL, (magma_int_t) nchrom, 
                                      NULL, NULL, (magma_int_t) nchrom, aux_work, -1, aux_iwork, -1, &info );
#endif
                    lwork   = (magma_int_t) aux_work[0];
                    liwork  = aux_iwork[0];

                    // allocate work arrays, eigenvalues and other stuff
                    w       = (user_real_t *)    malloc( nchrom       * sizeof(user_real_t)); if ( w == NULL ) MALLOC_ERR;
                    int Merr;
                    Merr = magma_imalloc_cpu   ( &iwork, liwork ); CHK_MERR; 
#ifdef USE_DOUBLES
                    Merr = magma_dmalloc_pinned( &wA , nchrom2 ); CHK_MERR;
                    Merr = magma_dmalloc_pinned( &work , lwork  ); CHK_MERR;
#else
                    Merr = magma_smalloc_pinned( &wA , nchrom2 ); CHK_MERR;
                    Merr = magma_smalloc_pinned( &work , lwork  ); CHK_MERR;
#endif
                    SSYEVD_ALLOC_FLAG = 0;  // is allocated, so we won't need to do it again
                }

#ifdef USE_DOUBLES
                magma_dsyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, kappa_d, (magma_int_t) nchrom,
                                  w, wA, (magma_int_t) nchrom, work, lwork, iwork, liwork, &info );
#else
                magma_ssyevd_gpu( MagmaVec, MagmaUpper, (magma_int_t) nchrom, kappa_d, (magma_int_t) nchrom,
                                  w, wA, (magma_int_t) nchrom, work, lwork, iwork, liwork, &info );
#endif
                if ( info != 0 ){ printf("ERROR: magma_dsyevd_gpu returned info %lld.\n", info ); exit(EXIT_FAILURE);}

                // copy eigenvalues to device memory
                hipMemcpy( w_d    , w    , nchrom*sizeof(user_real_t), hipMemcpyHostToDevice );
            }

            // ***          Done with the Diagonalization       *** //
            // ---------------------------------------------------- //



            // ---------------------------------------------------- //
            // ***              The Spectral Density            *** //

            if ( currentFrame == 0 && SPECD_FLAG )
            {

                // project the transition dipole moments onto the eigenbasis
                // MU_d = kappa_d**T x mu_d 
#ifdef USE_DOUBLES
                magma_dgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, (magma_int_t) nchrom , mux_d, 1, 0.0, MUX_d, 1, queue);
                magma_dgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, (magma_int_t) nchrom, muy_d, 1, 0.0, MUY_d, 1, queue);
                magma_dgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, (magma_int_t) nchrom, muz_d, 1, 0.0, MUZ_d, 1, queue);
#else
                magma_sgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, (magma_int_t) nchrom , mux_d, 1, 0.0, MUX_d, 1, queue);
                magma_sgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, (magma_int_t) nchrom, muy_d, 1, 0.0, MUY_d, 1, queue);
                magma_sgemv( MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                             1.0, kappa_d, (magma_int_t) nchrom, muz_d, 1, 0.0, MUZ_d, 1, queue);
#endif

                // Initializee the temporary array for spectral density
                for (int i = 0; i < nomega; i++) tmpSw[i] = 0.0;

                // Copy relevant variables to device memory
                hipMemcpy( omega_d, omega, nomega*sizeof(user_real_t), hipMemcpyHostToDevice );
                hipMemcpy( Sw_d   , tmpSw, nomega*sizeof(user_real_t), hipMemcpyHostToDevice );

                // calculate the spectral density on the GPU and copy back to the CPU
                get_spectral_density <<<numBlocks,blockSize>>> ( w_d, MUX_d, MUY_d, MUZ_d, omega_d, Sw_d, nomega, nchrom, t1, avef );
                hipMemcpy( tmpSw, Sw_d, nomega*sizeof(user_real_t), hipMemcpyDeviceToHost );

                // Copy temporary to persistant to get average spectral density over samples
                for (int i = 0; i < nomega; i++ ) Sw[i] += tmpSw[i];
            }

            // ***           Done the Spectral Density          *** //
            // ---------------------------------------------------- //




            // ---------------------------------------------------- //
            // ***           Time Correlation Function          *** //

            // cast variables to complex to calculate time correlation function (which is complex)
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( kappa_d, ckappa_d, nchrom2);
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( mux_d  , cmux_d  , nchrom );
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muy_d  , cmuy_d  , nchrom );
            cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muz_d  , cmuz_d  , nchrom );


            // ---------------------------------------------------- //
            // ***           Calculate the F matrix             *** //

            if ( currentFrame == 0 )
            {
                // initialize the F matrix at t=0 to the unit matrix
                makeI <<<numBlocks,blockSize>>> ( F_d, nchrom );

                // set the transition dipole moment at t=0
                cast_to_complex_GPU <<<numBlocks,blockSize>>> ( mux_d  , cmux0_d  , nchrom );
                cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muy_d  , cmuy0_d  , nchrom );
                cast_to_complex_GPU <<<numBlocks,blockSize>>> ( muz_d  , cmuz0_d  , nchrom );
            }
            else
            {
                if ( ifintmeth == 0 )   // Integrate with exact diagonalization
                {
                    // build the propigator
                    Pinit <<<numBlocks,blockSize>>> ( prop_d, w_d, nchrom, dt );
#ifdef USE_DOUBLES
                    // ctmpmat_d = ckappa_d * prop_d
                    magma_zgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                                 (magma_int_t) nchrom, MAGMA_ONE, ckappa_d, (magma_int_t) nchrom, prop_d, 
                                 (magma_int_t) nchrom, MAGMA_ZERO, ctmpmat_d, (magma_int_t) nchrom, queue );

                    // prop_d = ctmpmat_d * ckappa_d **T 
                    magma_zgemm( MagmaNoTrans, MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                                 (magma_int_t) nchrom, MAGMA_ONE, ctmpmat_d, (magma_int_t) nchrom, ckappa_d, 
                                 (magma_int_t) nchrom, MAGMA_ZERO, prop_d, (magma_int_t) nchrom, queue );

                    // ctmpmat_d = prop_d * F
                    magma_zgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                                 (magma_int_t) nchrom, MAGMA_ONE, prop_d, (magma_int_t) nchrom, F_d, 
                                 (magma_int_t) nchrom, MAGMA_ZERO, ctmpmat_d, (magma_int_t) nchrom, queue );

                    // copy the F matrix back from the temporary variable to F_d
                    magma_zcopy( (magma_int_t) nchrom2, ctmpmat_d , 1, F_d, 1, queue );
#else
                    // ctmpmat_d = ckappa_d * prop_d
                    magma_cgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                                 (magma_int_t) nchrom, MAGMA_ONE, ckappa_d, (magma_int_t) nchrom, prop_d, 
                                 (magma_int_t) nchrom, MAGMA_ZERO, ctmpmat_d, (magma_int_t) nchrom, queue );

                    // prop_d = ctmpmat_d * ckappa_d **T 
                    magma_cgemm( MagmaNoTrans, MagmaTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                                 (magma_int_t) nchrom, MAGMA_ONE, ctmpmat_d, (magma_int_t) nchrom, ckappa_d, 
                                 (magma_int_t) nchrom, MAGMA_ZERO, prop_d, (magma_int_t) nchrom, queue );

                    // ctmpmat_d = prop_d * F
                    magma_cgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                                 (magma_int_t) nchrom, MAGMA_ONE, prop_d, (magma_int_t) nchrom, F_d, 
                                 (magma_int_t) nchrom, MAGMA_ZERO, ctmpmat_d, (magma_int_t) nchrom, queue );

                    // copy the F matrix back from the temporary variable to F_d
                    magma_ccopy( (magma_int_t) nchrom2, ctmpmat_d , 1, F_d, 1, queue );
#endif
                }
                else if ( ifintmeth == 1 ) // Integrate F with 4th order Adams-Bashfort
                {                          // Note: The kappa matrix is assumed to be time independent over this integration cycle of max_int_steps
                    // reset current order if at the begining of a sample
                    if ( currentFrame == 1 ) order_counter = 1;

                    for ( int i=0; i<(int) max_int_steps; i++ )// take multiple steps
                    {
#ifdef USE_DOUBLES
                        // find current dF/dt = iF(t+i)k(t)
                        magma_zgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                                    (magma_int_t) nchrom, MAGMA_MAKE(0.0,1.0), F_d, nchrom, ckappa_d, nchrom,
                                     MAGMA_ZERO, ctmpmat_d, nchrom, queue );

                        // For the first step use Euler since previous values are not available
                        if ( order_counter == 1 )
                        {
                            // save current value for later
                            magma_zcopy( (magma_int_t) nchrom2, ctmpmat_d , 1, k1_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE(dt/HBAR/max_int_steps,0), k1_d, 1, F_d, 1, queue );
                            order_counter = 2;
                        }
                        // use ADAMS bensforth two-step method for step 2
                        else if ( order_counter == 2 )
                        {
                            // save current values for later
                            magma_zcopy( (magma_int_t) nchrom2, ctmpmat_d , 1, k2_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE( 3.0/2.0*dt/HBAR/max_int_steps,0), k2_d, 1, F_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE(-1.0/2.0*dt/HBAR/max_int_steps,0), k1_d, 1, F_d, 1, queue );
                            order_counter = 3;
                        }
                        // use ADAMS bensforth three-step method
                        else if ( order_counter == 3 )
                        {
                            // save current values for later
                            magma_zcopy( (magma_int_t) nchrom2, ctmpmat_d , 1, k3_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE(23.0/12.0*dt/HBAR/max_int_steps,0), k3_d, 1, F_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE(-4.0/3.0 *dt/HBAR/max_int_steps,0), k2_d, 1, F_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE( 5.0/12.0*dt/HBAR/max_int_steps,0), k1_d, 1, F_d, 1, queue );
                            order_counter = 4;
                        }
                        // use ADAMS bensforth four-step method
                        else
                        {
                            // save current values for later
                            magma_zcopy( (magma_int_t) nchrom2, ctmpmat_d , 1, k4_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE( 55.0/24.0 *dt/HBAR/max_int_steps,0), k4_d, 1, F_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE(-59.0/24.0 *dt/HBAR/max_int_steps,0), k3_d, 1, F_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE( 37.0/24.0 *dt/HBAR/max_int_steps,0), k2_d, 1, F_d, 1, queue );
                            magma_zaxpy( (magma_int_t) nchrom2, MAGMA_MAKE(-3.0/8.0   *dt/HBAR/max_int_steps,0), k1_d, 1, F_d, 1, queue );
                            // shuffle definitions for next iteration //
                            magma_zcopy( (magma_int_t) nchrom2, k2_d, 1, k1_d, 1, queue );
                            magma_zcopy( (magma_int_t) nchrom2, k3_d, 1, k2_d, 1, queue );
                            magma_zcopy( (magma_int_t) nchrom2, k4_d, 1, k3_d, 1, queue );
                        }

#else
                        // find current dF/dt = iF(t+i)k(t)
                        magma_cgemm( MagmaNoTrans, MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, 
                                    (magma_int_t) nchrom, MAGMA_MAKE(0.0,1.0), F_d, nchrom, ckappa_d, nchrom,
                                     MAGMA_ZERO, ctmpmat_d, nchrom, queue );

                        // For the first step use Euler since previous values are not available
                        if ( order_counter == 1 )
                        {
                            // save current value for later
                            magma_ccopy( (magma_int_t) nchrom2, ctmpmat_d , 1, k1_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE(dt/HBAR/max_int_steps,0), k1_d, 1, F_d, 1, queue );
                            order_counter = 2;
                        }
                        // use ADAMS bensforth two-step method for step 2
                        else if ( order_counter == 2 )
                        {
                            // save current values for later
                            magma_ccopy( (magma_int_t) nchrom2, ctmpmat_d , 1, k2_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE( 3.0/2.0*dt/HBAR/max_int_steps,0), k2_d, 1, F_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE(-1.0/2.0*dt/HBAR/max_int_steps,0), k1_d, 1, F_d, 1, queue );
                            order_counter = 3;
                        }
                        // use ADAMS bensforth three-step method
                        else if ( order_counter == 3 )
                        {
                            // save current values for later
                            magma_ccopy( (magma_int_t) nchrom2, ctmpmat_d , 1, k3_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE(23.0/12.0*dt/HBAR/max_int_steps,0), k3_d, 1, F_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE(-4.0/3.0 *dt/HBAR/max_int_steps,0), k2_d, 1, F_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE( 5.0/12.0*dt/HBAR/max_int_steps,0), k1_d, 1, F_d, 1, queue );
                            order_counter = 4;
                        }
                        // use ADAMS bensforth four-step method
                        else
                        {
                            // save current values for later
                            magma_ccopy( (magma_int_t) nchrom2, ctmpmat_d , 1, k4_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE( 55.0/24.0 *dt/HBAR/max_int_steps,0), k4_d, 1, F_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE(-59.0/24.0 *dt/HBAR/max_int_steps,0), k3_d, 1, F_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE( 37.0/24.0 *dt/HBAR/max_int_steps,0), k2_d, 1, F_d, 1, queue );
                            magma_caxpy( (magma_int_t) nchrom2, MAGMA_MAKE(-3.0/8.0   *dt/HBAR/max_int_steps,0), k1_d, 1, F_d, 1, queue );
                            // shuffle definitions for next iteration //
                            magma_ccopy( (magma_int_t) nchrom2, k2_d, 1, k1_d, 1, queue );
                            magma_ccopy( (magma_int_t) nchrom2, k3_d, 1, k2_d, 1, queue );
                            magma_ccopy( (magma_int_t) nchrom2, k4_d, 1, k3_d, 1, queue );
                        }
#endif
                    }
                }
            }
            // ***           Done updating the F matrix         *** //


            // calculate mFm for x y and z components
            // tcfx = cmux0_d**T * F_d *cmux_d
#ifdef USE_DOUBLES
            // x
            magma_zgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_ONE, F_d, (magma_int_t) nchrom,
                         cmux0_d, 1, MAGMA_ZERO, tmpmu_d, 1, queue);
            tcfx = magma_zdotu( (magma_int_t) nchrom, cmux_d, 1, tmpmu_d, 1, queue );

            // y
            magma_zgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_ONE, F_d, (magma_int_t) nchrom,
                         cmuy0_d, 1, MAGMA_ZERO, tmpmu_d, 1, queue);
            tcfy = magma_zdotu( (magma_int_t) nchrom, cmuy_d, 1, tmpmu_d, 1, queue );

            // z
            magma_zgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_ONE, F_d, (magma_int_t) nchrom,
                         cmuz0_d, 1, MAGMA_ZERO, tmpmu_d, 1, queue);
            tcfz = magma_zdotu( (magma_int_t) nchrom, cmuz_d, 1, tmpmu_d, 1, queue );
#else
            // x
            magma_cgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_C_ONE, F_d, (magma_int_t) nchrom,
                         cmux0_d, 1, MAGMA_C_ZERO, tmpmu_d, 1, queue);
            tcfx = magma_cdotu( (magma_int_t) nchrom, cmux_d, 1, tmpmu_d, 1, queue );

            // y
            magma_cgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_C_ONE, F_d, (magma_int_t) nchrom,
                         cmuy0_d, 1, MAGMA_C_ZERO, tmpmu_d, 1, queue);
            tcfy = magma_cdotu( (magma_int_t) nchrom, cmuy_d, 1, tmpmu_d, 1, queue );

            // z
            magma_cgemv( MagmaNoTrans, (magma_int_t) nchrom, (magma_int_t) nchrom, MAGMA_C_ONE, F_d, (magma_int_t) nchrom,
                         cmuz0_d, 1, MAGMA_C_ZERO, tmpmu_d, 1, queue);
            tcfz = magma_cdotu( (magma_int_t) nchrom, cmuz_d, 1, tmpmu_d, 1, queue );
#endif

            // accumulate the tcf over the samples
            tcftmp                = MAGMA_ADD( tcfx  , tcfy );
            tcftmp                = MAGMA_ADD( tcftmp, tcfz );
            tcf[ currentFrame ]   = MAGMA_ADD( tcf[currentFrame], tcftmp );

            // ***        Done with Time Correlation            *** //
            // ---------------------------------------------------- //


            // update progress bar if simulation is big enough, otherwise it really isn't necessary
            if ( nchrom > 400 && !interrupted )
            {
                printProgress( currentFrame, ntcfpoints-1 );
            }
            
            // done with current frame, move to next
            currentFrame += 1;
        }

        // done with current sample, move to next, and reset currentFrame to 0
        currentSample +=1;
        currentFrame  = 0;

        checkpoint( argv, gmxf, cptf, outf, model, &ifintmeth, &dt, &ntcfpoints, &nsamples, &sampleEvery, &t1, 
                    &avef, &omegaStart, &omegaStop, &omegaStep, &natom_mol, &nchrom_mol, &nzeros, &beginTime,
                    &SPECD_FLAG, &max_int_steps, nchrom, nomega, &currentSample, &currentFrame, tcf, Sw, F_d, 
                    cmux0_d, cmuy0_d, cmuz0_d, CP_WRITE );

        }
    } // end outer loop


    printf("\n\n----------------------------------------------------------\n");
    printf("Finishing up...\n");

    // close xdr file
    xdrfile_close(trj);


    // pad the time correlation function with zeros, copy to device memory and perform fft
    // fourier transform the time correlation function on the GPU
    pdtcf = (user_complex_t *) calloc( ntcfpoints+nzeros, sizeof(user_complex_t));
    for ( int i = 0; i < ntcfpoints; i++ )
    {
        // multiply the tcf by the relaxation term
        dcy      = MAGMA_MAKE(exp( -1.0 * i * dt / ( 2.0 * t1 )), 0.0);
        tcf[i]   = MAGMA_MUL( tcf[i], dcy );
        pdtcf[i] = MAGMA_DIV(tcf[i], MAGMA_MAKE( nsamples, 0.0 ));

    }
    for ( int i = 0; i < nzeros; i++ )
    {
        pdtcf[i+ntcfpoints] = MAGMA_ZERO;
    }

    hipMalloc( &pdtcf_d  , (ntcfpoints+nzeros)*sizeof(user_complex_t));
    hipMemcpy( pdtcf_d, pdtcf, (ntcfpoints+nzeros)*sizeof(user_complex_t), hipMemcpyHostToDevice );

#ifdef USE_DOUBLES
    hipfftPlan1d  ( &plan, ntcfpoints+nzeros, HIPFFT_Z2D, 1);
    hipfftExecZ2D ( plan, pdtcf_d, Ftcf_d );
#else
    hipfftPlan1d  ( &plan, ntcfpoints+nzeros, HIPFFT_C2R, 1);
    hipfftExecC2R ( plan, pdtcf_d, Ftcf_d );
#endif
    hipMemcpy   ( Ftcf, Ftcf_d, ntcfpointsR*sizeof(user_real_t), hipMemcpyDeviceToHost );
    hipfftDestroy(plan);


    // normalize spectra by number of samples
    for ( int i = 0; i < ntcfpointsR; i++ )
    {
        Ftcf[i] = Ftcf[i] ;/// (user_real_t) nsamples; 
    }
    if ( SPECD_FLAG )
    {
        for ( int i = 0; i < nomega; i++)
        {
            Sw[i]   = Sw[i] / (user_real_t) nsamples;
        }
    }

    // set base name for output files
    char * fname;
    fname = (char *) malloc( strlen(outf) + 9 );

    // write time correlation function
    rtcf = fopen(strcat(strcpy(fname,outf),"rtcf.dat"), "w");
    itcf = fopen(strcat(strcpy(fname,outf),"itcf.dat"), "w");
    for ( int i = 0; i < ntcfpoints; i++ )
    {
        fprintf( rtcf, "%g %g \n", i*dt, MAGMA_REAL( tcf[i] ) );
        fprintf( itcf, "%g %g \n", i*dt, MAGMA_IMAG( tcf[i] ) );
    }
    fclose( rtcf );
    fclose( itcf );

    // write the spectral density to file
    if ( SPECD_FLAG )
    {
        spec_density = fopen(strcat(strcpy(fname,outf),"spdn.dat"), "w");
        for ( int i = 0; i < nomega; i++)
        {
            fprintf(spec_density, "%g %g\n", omega[i], Sw[i]);
        }
        fclose(spec_density);
    }

    // Write the absorption lineshape... Since the C2R transform is inverse by default, the frequencies have to be negated
    // note if you need to compare with YICUN's code, divide Ftcf by 2
    spec_lineshape = fopen(strcat(strcpy(fname,outf),"spec.dat"),"w");
    factor         = 2*PI*HBAR/(dt*(ntcfpoints+nzeros));          // conversion factor to give energy and correct intensity from FFT
    for ( int i = (ntcfpoints+nzeros)/2; i < ntcfpoints+nzeros; i++ )   // "negative" FFT frequencies
    {
        if ( -1*(i-ntcfpoints-nzeros)*factor + avef <= (user_real_t) omegaStop  )
        {
            fprintf(spec_lineshape, "%g %g\n", -1*(i-ntcfpoints-nzeros)*factor + avef, Ftcf[i]/(factor*(ntcfpoints+nzeros)));// TO COMPARE WITH YICUN
        }
    }
    for ( int i = 0; i < ntcfpoints+nzeros / 2 ; i++)                   // "positive" FFT frequencies
    {
        if ( -1*i*factor + avef >= (user_real_t) omegaStart)
        {
            fprintf(spec_lineshape, "%g %g\n", -1*i*factor + avef, Ftcf[i]/(factor*(ntcfpoints+nzeros)));// TO COMPARE WITH YICUN
        }
    }
    fclose(spec_lineshape);

    // free memory on the CPU and GPU and finalize magma library
    magma_queue_destroy( queue );

    free(x);
    free(Ftcf);
    free(tcf);
    free(pdtcf);

    hipFree(x_d);
    hipFree(Ftcf_d);
    hipFree(mux_d); 
    hipFree(muy_d);
    hipFree(muz_d);
    hipFree(eproj_d);
    hipFree(kappa_d);
    hipFree(cmux_d); 
    hipFree(cmuy_d);
    hipFree(cmuz_d);
    hipFree(cmux0_d); 
    hipFree(cmuy0_d);
    hipFree(cmuz0_d);
    hipFree(tmpmu_d);
    hipFree(ckappa_d); 
    hipFree(F_d);
    hipFree(ctmpmat_d);
    hipFree(tcf_d);

    magma_free(pdtcf_d);


    // free memory used for diagonalization
    if ( SSYEVD_ALLOC_FLAG == 0 )
    {
        free(w);
        free(iwork);
        magma_free_pinned( work );
        magma_free_pinned( wA );
    }

    // free memory used in spectral density calculation
    if ( SPECD_FLAG ) // only used if the spetral density is calculated
    {
        // CPU arrays
        free(omega);
        free(Sw);
        free(tmpSw);

        // GPU arrays
        hipFree(MUX_d); 
        hipFree(MUY_d);
        hipFree(MUZ_d);
        hipFree(omega_d);
        hipFree(Sw_d);
        hipFree(w_d);
    }
 
    // free memory for integration of F depending on which method is used
    if ( ifintmeth == 0 ) // only used for the exact integration method
    {
        hipFree(prop_d);
    }
    else if ( ifintmeth == 1 ) // only used for adams integration method
    {
        hipFree(k1_d);
        hipFree(k2_d);
        hipFree(k3_d);
        hipFree(k4_d);
    }



    // final call to finalize magma math library
    magma_finalize();

    end = time(NULL);
    printf("\n>>> Done with the calculation in %f seconds.\n", difftime(end,start));

    return 0;
}

/**********************************************************
   
   BUILD ELECTRIC FIELD PROJECTION ALONG OH BONDS
                    GPU FUNCTION

 **********************************************************/
__global__
void get_eproj_GPU( rvec *x, float boxl, int natoms, int natom_mol, int nchrom, int nchrom_mol, int nmol, int model, user_real_t  *eproj )
{
    
    int n, m, i, j, istart, istride;
    int chrom;
    user_real_t mox[DIM];                     // oxygen position on molecule m
    user_real_t mx[DIM];                      // atom position on molecule m
    user_real_t nhx[DIM];                     // hydrogen position on molecule n of the current chromophore
    user_real_t nox[DIM];                     // oxygen position on molecule n
    user_real_t nohx[DIM];                    // the unit vector pointing along the OH bond for the current chromophore
    user_real_t mom[DIM];                     // the OM vector on molecule m
    user_real_t dr[DIM];                      // the min image vector between two atoms
    user_real_t r;                            // the distance between two atoms 
    const float cutoff = 0.7831;         // the oh cutoff distance
    const float bohr_nm = 18.8973;       // convert from bohr to nanometer
    user_real_t efield[DIM];                  // the electric field vector

    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // Loop over the chromophores belonging to the current thread
    for ( chrom = istart; chrom < nchrom; chrom += istride )
    {
        // calculate the molecule hosting the current chromophore 
        n = chrom / nchrom_mol;

        // initialize the electric field vector to zero at this chromophore
        efield[0]   =   0.;
        efield[1]   =   0.;
        efield[2]   =   0.;


        // ***          GET INFO ABOUT MOLECULE N HOSTING CHROMOPHORE       *** //
        //                      N IS OUR REFERENCE MOLECULE                     //
        // get the position of the hydrogen associated with the current stretch 
        // NOTE: I'm making some assumptions about the ordering of the positions, 
        // this can be changed if necessary for a more robust program
        // Throughout, I assume that the atoms are grouped into molecules and that
        // every 4th molecule starting at 0 (1, 2, 3) is OW (HW1, HW2, MW)
        if ( chrom % 2 == 0 ){      //HW1
            nhx[0]  = x[ n*natom_mol + 1 ][0];
            nhx[1]  = x[ n*natom_mol + 1 ][1];
            nhx[2]  = x[ n*natom_mol + 1 ][2];
        }
        else if ( chrom % 2 == 1 ){ //HW2
            nhx[0]  = x[ n*natom_mol + 2 ][0];
            nhx[1]  = x[ n*natom_mol + 2 ][1];
            nhx[2]  = x[ n*natom_mol + 2 ][2];
        }

        // The oxygen position
        nox[0]  = x[ n*natom_mol ][0];
        nox[1]  = x[ n*natom_mol ][1];
        nox[2]  = x[ n*natom_mol ][2];

        // The oh unit vector
        nohx[0] = minImage( nhx[0] - nox[0], boxl );
        nohx[1] = minImage( nhx[1] - nox[1], boxl );
        nohx[2] = minImage( nhx[2] - nox[2], boxl );
        r       = mag3(nohx);
        nohx[0] /= r;
        nohx[1] /= r;
        nohx[2] /= r;

        // for testing with YICUN -- can change to ROH later...
        //nohx[0] /= 0.09572;
        //nohx[1] /= 0.09572;
        //nohx[2] /= 0.09572;
 
        // ***          DONE WITH MOLECULE N                                *** //



        // ***          LOOP OVER ALL OTHER MOLECULES                       *** //
        for ( m = 0; m < nmol; m++ ){

            // skip the reference molecule
            if ( m == n ) continue;

            // get oxygen position on current molecule
            mox[0] = x[ m*natom_mol ][0];
            mox[1] = x[ m*natom_mol ][1];
            mox[2] = x[ m*natom_mol ][2];

            // find displacement between oxygen on m and hydrogen on n
            dr[0]  = minImage( mox[0] - nhx[0], boxl );
            dr[1]  = minImage( mox[1] - nhx[1], boxl );
            dr[2]  = minImage( mox[2] - nhx[2], boxl );
            r      = mag3(dr);

            // skip if the distance is greater than the cutoff
            if ( r > cutoff ) continue;

            // loop over all atoms in the current molecule and calculate the electric field 
            // (excluding the oxygen atoms since they have no charge)
            for ( i=1; i < natom_mol; i++ ){

                // position of current atom
                mx[0] = x[ m*natom_mol + i ][0];
                mx[1] = x[ m*natom_mol + i ][1];
                mx[2] = x[ m*natom_mol + i ][2];

                // Move m site to TIP4P distance if model is E3B3 or TIP4P2005 -- this must be done to use the TIP4P map
                if ( i == 3 )
                {
                    if ( model != 0 ) 
                    {
                        // get the OM unit vector
                        mom[0] = minImage( mx[0] - mox[0], boxl);
                        mom[1] = minImage( mx[1] - mox[1], boxl);
                        mom[2] = minImage( mx[2] - mox[2], boxl);
                        r      = mag3(mom);

                        // TIP4P OM distance is 0.015 nm along the OM bond
                        mx[0] = mox[0] + 0.0150*mom[0]/r;
                        mx[1] = mox[1] + 0.0150*mom[1]/r;
                        mx[2] = mox[2] + 0.0150*mom[2]/r;
                    }
                }

                // the minimum image displacement between the reference hydrogen and the current atom
                // NOTE: this converted to bohr so the efield will be in au
                dr[0]  = minImage( nhx[0] - mx[0], boxl )*bohr_nm;
                dr[1]  = minImage( nhx[1] - mx[1], boxl )*bohr_nm;
                dr[2]  = minImage( nhx[2] - mx[2], boxl )*bohr_nm;
                r      = mag3(dr);

                // Add the contribution of the current atom to the electric field
                if ( i < 3  ){              // HW1 and HW2
                    for ( j=0; j < DIM; j++){
                        efield[j] += 0.52 * dr[j] / (r*r*r);
                    }
                }
                else if ( i == 3 ){         // MW (note the negative sign)
                    for ( j=0; j < DIM; j++){
                        efield[j] -= 1.04 * dr[j] / (r*r*r);
                    }
                }
            } // end loop over atoms in molecule m

        } // end loop over molecules m

        // project the efield along the OH bond to get the relevant value for the map
        eproj[chrom] = dot3( efield, nohx );

    } // end loop over reference chromophores
}

/**********************************************************
   
   BUILD HAMILTONIAN AND RETURN TRANSITION DIPOLE VECTOR
    FOR EACH CHROMOPHORE ON THE GPU

 **********************************************************/
__global__
void get_kappa_GPU( rvec *x, float boxl, int natoms, int natom_mol, int nchrom, int nchrom_mol, int nmol, 
                    user_real_t *eproj, user_real_t *kappa, user_real_t *mux, user_real_t *muy, user_real_t *muz, user_real_t avef)
{
    
    int n, m, istart, istride;
    int chromn, chromm;
    user_real_t mox[DIM];                         // oxygen position on molecule m
    user_real_t mhx[DIM];                         // atom position on molecule m
    user_real_t nhx[DIM];                         // hydrogen position on molecule n of the current chromophore
    user_real_t nox[DIM];                         // oxygen position on molecule n
    user_real_t noh[DIM];
    user_real_t moh[DIM];
    user_real_t nmu[DIM];
    user_real_t mmu[DIM];
    user_real_t mmuprime;
    user_real_t nmuprime;
    user_real_t dr[DIM];                          // the min image vector between two atoms
    user_real_t r;                                // the distance between two atoms 
    const user_real_t bohr_nm    = 18.8973;       // convert from bohr to nanometer
    const user_real_t cm_hartree = 2.1947463E5;   // convert from cm-1 to hartree
    user_real_t En, Em;                           // the electric field projection
    user_real_t xn, xm, pn, pm;                   // the x and p from the map
    user_real_t wn, wm;                           // the energies

    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // Loop over the chromophores belonging to the current thread and fill in kappa for that row
    for ( chromn = istart; chromn < nchrom; chromn += istride )
    {
        // calculate the molecule hosting the current chromophore 
        // and get the corresponding electric field at the relevant hydrogen
        n   = chromn / nchrom_mol;
        En  = eproj[chromn];

        // build the map
        wn  = 3760.2 - 3541.7*En - 152677.0*En*En;
        xn  = 0.19285 - 1.7261E-5 * wn;
        pn  = 1.6466  + 5.7692E-4 * wn;
        nmuprime = 0.1646 + 11.39*En + 63.41*En*En;

        // and calculate the location of the transition dipole moment
        // SEE calc_efield_GPU for assumptions about ordering of atoms
        nox[0]  = x[ n*natom_mol ][0];
        nox[1]  = x[ n*natom_mol ][1];
        nox[2]  = x[ n*natom_mol ][2];
        if ( chromn % 2 == 0 )       //HW1
        {
            nhx[0]  = x[ n*natom_mol + 1 ][0];
            nhx[1]  = x[ n*natom_mol + 1 ][1];
            nhx[2]  = x[ n*natom_mol + 1 ][2];
        }
        else if ( chromn % 2 == 1 )  //HW2
        {
            nhx[0]  = x[ n*natom_mol + 2 ][0];
            nhx[1]  = x[ n*natom_mol + 2 ][1];
            nhx[2]  = x[ n*natom_mol + 2 ][2];
        }

        // The OH unit vector
        noh[0] = minImage( nhx[0] - nox[0], boxl );
        noh[1] = minImage( nhx[1] - nox[1], boxl );
        noh[2] = minImage( nhx[2] - nox[2], boxl );
        r      = mag3(noh);
        noh[0] /= r;
        noh[1] /= r;
        noh[2] /= r;

        // The location of the TDM
        nmu[0] = minImage( nox[0] + 0.067 * noh[0], boxl );
        nmu[1] = minImage( nox[1] + 0.067 * noh[1], boxl );
        nmu[2] = minImage( nox[2] + 0.067 * noh[2], boxl );
        
        // and the TDM vector to return
        mux[chromn] = noh[0] * nmuprime * xn;
        muy[chromn] = noh[1] * nmuprime * xn;
        muz[chromn] = noh[2] * nmuprime * xn;



        // Loop over all other chromophores
        for ( chromm = 0; chromm < nchrom; chromm ++ )
        {
            // calculate the molecule hosting the current chromophore 
            // and get the corresponding electric field at the relevant hydrogen
            m   = chromm / nchrom_mol;
            Em  = eproj[chromm];

            // also get the relevent x and p from the map
            wm  = 3760.2 - 3541.7*Em - 152677.0*Em*Em;
            xm  = 0.19285 - 1.7261E-5 * wm;
            pm  = 1.6466  + 5.7692E-4 * wm;
            mmuprime = 0.1646 + 11.39*Em + 63.41*Em*Em;

            // the diagonal energy
            if ( chromn == chromm )
            {
                // Note that this is a flattened 2d array -- subtract high frequency energies to get rid of highly oscillatory parts of the F matrix
                kappa[chromn*nchrom + chromm]   = wm - avef;
            }

            // intramolecular coupling
            else if ( m == n )
            {
                kappa[chromn*nchrom + chromm]   =  (-1361.0 + 27165*(En + Em))*xn*xm - 1.887*pn*pm;
            }

            // intermolecular coupling
            else
            {
                
                // calculate the distance between dipoles
                // they are located 0.67 A from the oxygen along the OH bond
                // tdm position on chromophore n
                mox[0]  = x[ m*natom_mol ][0];
                mox[1]  = x[ m*natom_mol ][1];
                mox[2]  = x[ m*natom_mol ][2];
                if ( chromm % 2 == 0 )       //HW1
                {
                    mhx[0]  = x[ m*natom_mol + 1 ][0];
                    mhx[1]  = x[ m*natom_mol + 1 ][1];
                    mhx[2]  = x[ m*natom_mol + 1 ][2];
                }
                else if ( chromm % 2 == 1 )  //HW2
                {
                    mhx[0]  = x[ m*natom_mol + 2 ][0];
                    mhx[1]  = x[ m*natom_mol + 2 ][1];
                    mhx[2]  = x[ m*natom_mol + 2 ][2];
                }

                // The OH unit vector
                moh[0] = minImage( mhx[0] - mox[0], boxl );
                moh[1] = minImage( mhx[1] - mox[1], boxl );
                moh[2] = minImage( mhx[2] - mox[2], boxl );
                r      = mag3(moh);
                moh[0] /= r;
                moh[1] /= r;
                moh[2] /= r;

                // The location of the TDM and the dipole derivative
                mmu[0] = minImage( mox[0] + 0.067 * moh[0], boxl );
                mmu[1] = minImage( mox[1] + 0.067 * moh[1], boxl );
                mmu[2] = minImage( mox[2] + 0.067 * moh[2], boxl );

                // the distance between TDM on N and on M and convert to unit vector
                dr[0] = minImage( nmu[0] - mmu[0], boxl );
                dr[1] = minImage( nmu[1] - mmu[1], boxl );
                dr[2] = minImage( nmu[2] - mmu[2], boxl );
                r     = mag3( dr );
                dr[0] /= r;
                dr[1] /= r;
                dr[2] /= r;
                r     *= bohr_nm; // convert to bohr

                // The coupling in the transition dipole approximation in wavenumber
                // Note the conversion to wavenumber
                kappa[chromn*nchrom + chromm]   = ( dot3( noh, moh ) - 3.0 * dot3( noh, dr ) * 
                                                    dot3( moh, dr ) ) / ( r*r*r ) * 
                                                    xn*xm*nmuprime*mmuprime*cm_hartree;
            }// end intramolecular coupling
        }// end loop over chromm
    }// end loop over reference
}


/**********************************************************
   
        Calculate the Spectral Density

 **********************************************************/
__global__
void get_spectral_density( user_real_t *w, user_real_t *MUX, user_real_t *MUY, user_real_t *MUZ, user_real_t *omega, user_real_t *Sw, 
                           int nomega, int nchrom, user_real_t t1, user_real_t avef ){

    int istart, istride, i, chromn;
    user_real_t wi, dw, MU2, gamma;
    
    // split up each desired frequency to separate thread on GPU
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // the linewidth parameter
    gamma = HBAR/(t1 * 2.0);

    // Loop over the chromophores belonging to the current thread and fill in kappa for that row
    for ( i = istart; i < nomega; i += istride )
    {
        // get current frequency
        wi = omega[i];
        
        // Loop over all chromophores calculatint the spectral intensity at the current frequency
        for ( chromn = 0; chromn < nchrom; chromn ++ ){
            // calculate the TDM squared and get the mode energy
            MU2     = MUX[chromn]*MUX[chromn] + MUY[chromn]*MUY[chromn] + MUZ[chromn]*MUZ[chromn];
            dw      = wi - (w[chromn] + avef) ; // also adjust for avef subtracted from kappa

            // add a lorentzian lineshape to the spectral density
            Sw[i] += MU2 * gamma / ( dw*dw + gamma*gamma )/PI;
        }
    }
}

/**********************************************************
   
        HELPER FUNCTIONS FOR GPU CALCULATIONS
            CALLABLE FROM CPU AND GPU

 **********************************************************/



// The minimage image of a scalar
user_real_t minImage( user_real_t dx, user_real_t boxl )
{
    return dx - boxl*round(dx/boxl);
}



// The magnitude of a 3 dimensional vector
user_real_t mag3( user_real_t dx[3] )
{
    return sqrt( dot3( dx, dx ) );
}



// The dot product of a 3 dimensional vector
user_real_t dot3( user_real_t x[3], user_real_t y[3] )
{
    return  x[0]*y[0] + x[1]*y[1] + x[2]*y[2];
}



// cast the matrix from float to complex -- this may not be the best way to do this, but it is quick to implement
__global__
void cast_to_complex_GPU ( user_real_t *d_d, user_complex_t *z_d, magma_int_t n )
{
    int istart, istride, i;
    
    // split up each desired frequency to separate thread on GPU
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // convert from float to complex
    for ( i = istart; i < n; i += istride )
    {
        z_d[i] = MAGMA_MAKE( d_d[i], 0.0 ); 
    }
}

// initialize the propigation matrix
__global__
void Pinit ( user_complex_t *prop_d, user_real_t *w_d, int n, user_real_t dt )
{
    int istart, istride, i, j;
    user_real_t arg;
    
    // each will occour on a separate thread on the gpu
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    for ( i = istart; i < n; i += istride )
    {
        // zero matrix
        for ( j = 0; j < n; j ++ ) prop_d[ i*n + j] = MAGMA_ZERO;
        // P = exp(iwt/hbar)
        arg   = w_d[i] * dt / HBAR;
        prop_d[ i*n + i ] = MAGMA_MAKE( cos(arg), sin(arg) );
    }
}


// initialize the F matrix on the gpu to the unit matrix
__global__
void makeI ( user_complex_t *mat, int n )
{
    int istart, istride, i, j;
    
    // each will occour on a separate thread on the gpu
    istart  =   blockIdx.x * blockDim.x + threadIdx.x;
    istride =   blockDim.x * gridDim.x;

    // convert from float to complex
    for ( i = istart; i < n; i += istride )
    {
        for ( j = 0; j < n; j++ ) mat[ i*n + j ] = MAGMA_ZERO;
        mat[ i * n + i ] = MAGMA_ONE;
    }
}


// parse input file to setup calculation
void ir_init( char *argv[], char gmxf[], char cptf[], char outf[], char model[], int *ifintmeth, user_real_t *dt, int *ntcfpoints, 
              int *nsamples, int *sampleEvery, user_real_t *t1, user_real_t *avef, int *omegaStart, int *omegaStop, 
              int *omegaStep, int *natom_mol, int *nchrom_mol, int *nzeros, user_real_t *beginTime, int *SPECD_FLAG,
              user_real_t *max_int_steps)
{
    char                para[MAX_STR_LEN];
    char                value[MAX_STR_LEN];

    FILE *inpf = fopen(argv[1],"r");
    if ( inpf == NULL )
    {
        printf("ERROR: Could not open %s. The first argument should contain  a  vaild\nfile name that points to a file containing the simulation parameters\n or a checkpoint file ending in '.cpt' to restart the simulation.", argv[1]);
        exit(EXIT_FAILURE);
    }
    else printf(">>> Reading parameters from input file %s\n", argv[1]);

    // Parse input file
    while (fscanf( inpf, "%s%s%*[^\n]", para, value ) != EOF)
    {
        if ( strcmp(para,"xtcf") == 0 ) 
        {
            sscanf( value, "%s", gmxf );
        }
        else if ( strcmp(para,"outf") == 0 )
        {
            sscanf( value, "%s", outf );
        }
        else if ( strcmp(para,"cptf") == 0 ) 
        {
            sscanf( value, "%s", cptf );
        }
        else if ( strcmp(para,"model") == 0 )
        {
            sscanf( value, "%s", model );
        }
        else if ( strcmp(para,"fintmeth") == 0 )
        {
            sscanf( value, "%d", (int *) ifintmeth );
            if ( *ifintmeth < 0 || *ifintmeth > 1 ) *ifintmeth = 0;
        }
        else if ( strcmp(para,"ntcfpoints") == 0 )
        {
            sscanf( value, "%d", (int *) ntcfpoints );
        }
        else if ( strcmp(para,"nsamples") == 0 )
        {
            sscanf( value, "%d", (int *) nsamples);
        }
        else if ( strcmp(para,"sampleEvery") == 0 )
        {
            sscanf( value, "%d", (int *) sampleEvery );
        }
        else if ( strcmp(para,"omegaStart") == 0 )
        {
            sscanf( value, "%d", (int *) omegaStart );
        }
        else if ( strcmp(para,"omegaStop") == 0 )
        {
            sscanf( value, "%d", (int *) omegaStop );
        }
        else if ( strcmp(para,"omegaStep") == 0 )
        {
            sscanf( value, "%d", (int *) omegaStep );
        }
        else if ( strcmp(para,"natom_mol") == 0 )
        {
            sscanf( value, "%d", (int *) natom_mol );
        }
        else if ( strcmp(para,"nchrom_mol") == 0 )
        {
            sscanf( value, "%d", (int *) nchrom_mol );
        }
        else if ( strcmp(para,"nzeros") == 0 )
        {
            sscanf( value, "%d", (int *) nzeros );
        }
        else if ( strcmp(para,"SPECD_FLAG") == 0 )
        {
            sscanf( value, "%d", (int *) SPECD_FLAG);
        }
#ifdef USE_DOUBLES
        else if ( strcmp(para,"dt") == 0 )
        {
            sscanf( value, "%lf", dt );
        }
        else if ( strcmp(para,"t1") == 0 )
        {
            sscanf( value, "%lf", t1 );
        }
        else if ( strcmp(para,"avef") == 0 )
        {
            sscanf( value, "%lf", avef );
        }
        else if ( strcmp(para,"beginTime") == 0 )
        {
            sscanf( value, "%lf", beginTime );
        }
        else if ( strcmp(para,"max_int_steps") == 0 )
        {
            sscanf( value, "%lf", max_int_steps);
        }
#else
        else if ( strcmp(para,"dt") == 0 )
        {
            sscanf( value, "%f", dt );
        }
        else if ( strcmp(para,"t1") == 0 )
        {
            sscanf( value, "%f", t1 );
        }
        else if ( strcmp(para,"avef") == 0 )
        {
            sscanf( value, "%f", avef );
        }
        else if ( strcmp(para,"beginTime") == 0 )
        {
            sscanf( value, "%f", beginTime );
        }
        else if ( strcmp(para,"max_int_steps") == 0 )
        {
            sscanf( value, "%f", max_int_steps);
        }
#endif
        else
        {
            printf("WARNING: Parameter %s in input file %s not recognized, ignoring.\n", para, argv[1]);
        }
    }

    fclose(inpf);
    printf(">>> Done reading input file and setting parameters\n");

}



// Progress bar to keep updated on tcf
void printProgress( int currentStep, int totalSteps )
{
    user_real_t percentage = (user_real_t) currentStep / (user_real_t) totalSteps;
    int lpad = (int) (percentage*PWID);
    int rpad = PWID - lpad;
    fprintf(stderr, "\r [%.*s%*s]%3d%%", lpad, PSTR, rpad, "",(int) (percentage*100));
}



// Checkpoint the simulation
// TODO: Things that don't need to be consistent -- t1, nzeros, nsamples, omega
void checkpoint( char *argv[], char gmxf[], char cptf[], char outf[], char model[], int *ifintmeth, user_real_t *dt, int *ntcfpoints, 
                 int *nsamples, int *sampleEvery, user_real_t *t1, user_real_t *avef, int *omegaStart, int *omegaStop, int *omegaStep,
                 int *natom_mol, int *nchrom_mol, int *nzeros, user_real_t *beginTime, int *SPECD_FLAG, user_real_t *max_int_steps, int nchrom, int nomega,
                 int *currentSample, int *currentFrame, user_complex_t *tcf, user_real_t *Sw, user_complex_t *F_d, user_complex_t *cmux0_d,
                 user_complex_t *cmuy0_d, user_complex_t *cmuz0_d, int RWI_FLAG )
{

    FILE *cptfp;                // checkpoint file pointer
    char bakf[MAX_STR_LEN];     // backup file name
    user_complex_t *tmparr;     // temporary variable to transfer variables from CPU <-> GPU for reading/writing
 

    // Writing the checkpoint file
    if ( RWI_FLAG == CP_WRITE )
    {
        // if cpt file exists, back it up before proceeding
        sprintf(bakf,"%s.bak",cptf);
        if( access( cptf, F_OK ) != -1 ) rename( cptf, bakf );

        // back up calculation
        cptfp = fopen(cptf, "wb");

        // Write the simulation parameters      
        fwrite( gmxf        , MAX_STR_LEN           , 1, cptfp );         // trajectory file
        fwrite( cptf        , MAX_STR_LEN           , 1, cptfp );         // checkpoint file
        fwrite( outf        , MAX_STR_LEN           , 1, cptfp );         // output file names
        fwrite( model       , MAX_STR_LEN           , 1, cptfp );         // model
        fwrite( ifintmeth   , sizeof(int)           , 1, cptfp );         // integration method
        fwrite( ntcfpoints  , sizeof(int)           , 1, cptfp );         // number of tcf points
        fwrite( nsamples    , sizeof(int)           , 1, cptfp );         // number of samples -- TODO: Doesn't need to be the same
        fwrite( sampleEvery , sizeof(int)           , 1, cptfp );         // time between samples
        fwrite( omegaStart  , sizeof(int)           , 1, cptfp );         // omegaStart for spectral density
        fwrite( omegaStop   , sizeof(int)           , 1, cptfp );         // omegaStop  for spectral density
        fwrite( omegaStep   , sizeof(int)           , 1, cptfp );         // omegaStep  for spectral density
        fwrite( natom_mol   , sizeof(int)           , 1, cptfp );         // atoms per molecule
        fwrite( nchrom_mol  , sizeof(int)           , 1, cptfp );         // chromophores per molecule
        fwrite( nzeros      , sizeof(int)           , 1, cptfp );         // number of zeros to pad the tcf before FT -- TODO: Doesn't need to be the same
        fwrite( SPECD_FLAG  , sizeof(int)           , 1, cptfp );         // switch to calculate spectral density

        fwrite( max_int_steps, sizeof(user_real_t)  , 1, cptfp );         // max integration steps if using adams/bashforth integration
        fwrite( t1          , sizeof(user_real_t)   , 1, cptfp );         // relaxation time -- TODO: Doesn't need to be the same
        fwrite( dt          , sizeof(user_real_t)   , 1, cptfp );         // timestep
        fwrite( avef        , sizeof(user_real_t)   , 1, cptfp );         // average frequency
        fwrite( beginTime   , sizeof(user_real_t)   , 1, cptfp );         // time to start taking samples

        // Write the current configuration
        fwrite( currentSample,  sizeof(int)         , 1, cptfp );         // current sample number
        fwrite( currentFrame ,  sizeof(int)         , 1, cptfp );         // current frame  number
        fwrite( tcf, sizeof(user_complex_t), *ntcfpoints, cptfp );         // current time correlation function
        if ( *SPECD_FLAG ){
            fwrite( Sw,  sizeof(user_real_t),    nomega    , cptfp );     // current spectral density
        }
        if (*currentFrame !=0){// if at frame 0, this will all be generated on restart and doesn't need to be recorded
            // Malloc space for temporary variable for CPU <-> GPU transfer
            tmparr  = (user_complex_t *) malloc( nchrom*nchrom * sizeof(user_complex_t));

            // copy memory from gpu to cpu and write
            hipMemcpy( tmparr, F_d     , nchrom*nchrom*sizeof(user_complex_t), hipMemcpyDeviceToHost );
            fwrite( tmparr  , sizeof(user_complex_t), nchrom*nchrom , cptfp ); // current F matrix -- not needed at frame 0

            hipMemcpy( tmparr, cmux0_d , nchrom*sizeof(user_complex_t), hipMemcpyDeviceToHost );
            fwrite( tmparr  , sizeof(user_complex_t), nchrom        , cptfp ); // mu0 -- not needed at frame 0

            hipMemcpy( tmparr, cmuy0_d , nchrom*sizeof(user_complex_t), hipMemcpyDeviceToHost );
            fwrite( tmparr  , sizeof(user_complex_t), nchrom        , cptfp ); // mu0 -- not needed at frame 0

            hipMemcpy( tmparr, cmuz0_d , nchrom*sizeof(user_complex_t), hipMemcpyDeviceToHost );
            fwrite( tmparr  , sizeof(user_complex_t), nchrom        , cptfp ); // mu0 -- not needed at frame 0

            free( tmparr );
        }

        // close the file
        fclose(cptfp);
    }
    // Read the configuration from the checkpoint file
    else
    {
        // if cpt file exists, read it and restart calculation, else abort
        if( access( argv[1], F_OK ) != -1 ) 
        {
            // Initialize the simulation by reading parameters
            if ( RWI_FLAG == CP_INIT )
            {
                // open the file
                cptfp = fopen(argv[1],"rb");

                // Read the simulation parameters      
                fread( gmxf        , MAX_STR_LEN           , 1, cptfp );         // trajectory file
                fread( cptf        , MAX_STR_LEN           , 1, cptfp );         // checkpoint file
                fread( outf        , MAX_STR_LEN           , 1, cptfp );         // output file names
                fread( model       , MAX_STR_LEN           , 1, cptfp );         // model
                fread( ifintmeth   , sizeof(int)           , 1, cptfp );         // integration method
                fread( ntcfpoints  , sizeof(int)           , 1, cptfp );         // number of tcf points
                fread( nsamples    , sizeof(int)           , 1, cptfp );         // number of samples -- TODO: Doesn't need to be the same
                fread( sampleEvery , sizeof(int)           , 1, cptfp );         // time between samples
                fread( omegaStart  , sizeof(int)           , 1, cptfp );         // omegaStart for spectral density
                fread( omegaStop   , sizeof(int)           , 1, cptfp );         // omegaStop  for spectral density
                fread( omegaStep   , sizeof(int)           , 1, cptfp );         // omegaStep  for spectral density
                fread( natom_mol   , sizeof(int)           , 1, cptfp );         // atoms per molecule
                fread( nchrom_mol  , sizeof(int)           , 1, cptfp );         // chromophores per molecule
                fread( nzeros      , sizeof(int)           , 1, cptfp );         // number of zeros to pad the tcf before FT -- TODO: Doesn't need to be the same
                fread( SPECD_FLAG  , sizeof(int)           , 1, cptfp );         // switch to calculate spectral density

                fread( max_int_steps,sizeof(user_real_t)   , 1, cptfp );         // max integration steps if using adams/bashforth integration
                fread( t1          , sizeof(user_real_t)   , 1, cptfp );         // relaxation time -- TODO: Doesn't need to be the same
                fread( dt          , sizeof(user_real_t)   , 1, cptfp );         // timestep
                fread( avef        , sizeof(user_real_t)   , 1, cptfp );         // average frequency
                fread( beginTime   , sizeof(user_real_t)   , 1, cptfp );         // time to start taking samples

                // close the file 
                fclose(cptfp);

            }
            // Read the current state
            else if ( RWI_FLAG == CP_READ )
            {
                // open the file
                cptfp = fopen(argv[1],"rb");
    
                // skip bytes containing simulation parameters
                fseek( cptfp, 4*MAX_STR_LEN + 11 * sizeof(int) + 5 * sizeof(user_real_t), SEEK_SET );

                // Write the current configuration
                fread( currentSample,  sizeof(int)         , 1, cptfp );         // current sample number
                fread( currentFrame ,  sizeof(int)         , 1, cptfp );         // current frame  number
                fread( tcf, sizeof(user_complex_t), *ntcfpoints, cptfp );         // current time correlation function
                if ( *SPECD_FLAG ){
                    fread( Sw,  sizeof(user_real_t),    nomega    , cptfp );     // current spectral density
                }
                if (*currentFrame !=0){
                    // Malloc space for temporary variable for CPU <-> GPU transfer
                    tmparr  = (user_complex_t *) malloc( nchrom*nchrom *sizeof(user_complex_t));

                    // read and copy memory from cpu to gpu
                    fread( tmparr  , sizeof(user_complex_t), nchrom*nchrom , cptfp ); // current F matrix -- not needed at frame 0
                    hipMemcpy( F_d, tmparr     , nchrom*nchrom*sizeof(user_complex_t), hipMemcpyHostToDevice );

                    fread( tmparr  , sizeof(user_complex_t), nchrom        , cptfp ); // mu0 -- not needed at frame 0
                    hipMemcpy( cmux0_d, tmparr, nchrom*sizeof(user_complex_t), hipMemcpyHostToDevice );

                    fread( tmparr  , sizeof(user_complex_t), nchrom        , cptfp ); // mu0 -- not needed at frame 0
                    hipMemcpy( cmuy0_d, tmparr, nchrom*sizeof(user_complex_t), hipMemcpyHostToDevice );

                    fread( tmparr  , sizeof(user_complex_t), nchrom        , cptfp ); // mu0 -- not needed at frame 0
                    hipMemcpy( cmuz0_d, tmparr, nchrom*sizeof(user_complex_t), hipMemcpyHostToDevice );

                    free( tmparr );
                }

                // close the file
                fclose(cptfp);

                // print message to user about the restart
                printf(">>> Found checkpoint file %s.\n>>> Will restart the calculation from sample %d and frame %d.",cptf, *currentSample+1, *currentFrame);
 
            }
        }
        else
        {
            printf(">>> No cpt file found (looking for %s). Aborting \n", argv[1]);
            exit(EXIT_FAILURE);
        }
    }
}

void signal_handler( int sig )
{
    //... program has recieved some signal
    interrupted=true;
    fprintf(stderr, "\nRecieved signal. Will write checkpoint file and exit.\n");
}
